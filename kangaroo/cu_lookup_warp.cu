#include "hip/hip_runtime.h"
#include "cu_lookup_warp.h"

#include "launch_utils.h"
#include "sampling.h"

namespace roo
{

//////////////////////////////////////////////////////
// Create Matlab Lookup table
//////////////////////////////////////////////////////

__global__ void KernCreateMatlabLookupTable(
    Image<float2> lookup, float fu, float fv, float u0, float v0, float k1, float k2
) {
    const uint u = blockIdx.x*blockDim.x + threadIdx.x;
    const uint v = blockIdx.y*blockDim.y + threadIdx.y;

    const float pnu = (u-u0) / fu;
    const float pnv = (v-v0) / fv;
    const float r = sqrt(pnu*pnu + pnv*pnv);
    const float rr = r*r;
    const float rf = 1 + k1*rr + k2*rr*rr; // + k3*rr*rr*rr;

    lookup(u,v) = make_float2(
        (pnu*rf /*+ 2*p1*pn.x*pn.y + p2*(rr + 2*pn.x*pn.x)*/) * fu + u0,
        (pnv*rf /*+ p1*(rr + 2*pn.y*pn.y) + 2*p2*pn.x*pn.y*/) * fv + v0
    );
}

void CreateMatlabLookupTable(
    Image<float2> lookup, float fu, float fv, float u0, float v0, float k1, float k2
) {
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, lookup);
    KernCreateMatlabLookupTable<<<gridDim,blockDim>>>(lookup,fu,fv,u0,v0,k1,k2);
}

//////////////////////////////////////////////////////
// Create Matlab Lookup table applying homography
//////////////////////////////////////////////////////

__global__ void KernCreateMatlabLookupTable(
    Image<float2> lookup, float fu, float fv, float u0, float v0, float k1, float k2, Mat<float,9> H_on
) {
    const uint x = blockIdx.x*blockDim.x + threadIdx.x;
    const uint y = blockIdx.y*blockDim.y + threadIdx.y;

    // Apply homography H_on, moving New image to Original
    const float hdiv = H_on[6] * x + H_on[7] * y + H_on[8];
    const float u = (H_on[0] * x + H_on[1] * y + H_on[2]) / hdiv;
    const float v = (H_on[3] * x + H_on[4] * y + H_on[5]) / hdiv;

    // Apply distortion to achieve true image coordinates
    const float pnu = (u-u0) / fu;
    const float pnv = (v-v0) / fv;
    const float r = sqrt(pnu*pnu + pnv*pnv);
    const float rr = r*r;
    const float rf = 1 + k1*rr + k2*rr*rr; // + k3*rr*rr*rr;

    float2 pos = make_float2(
        (pnu*rf /*+ 2*p1*pn.x*pn.y + p2*(rr + 2*pn.x*pn.x)*/) * fu + u0,
        (pnv*rf /*+ p1*(rr + 2*pn.y*pn.y) + 2*p2*pn.x*pn.y*/) * fv + v0
    );

    // Clamp to image bounds
    pos.x = max(pos.x, 1.0f);
    pos.y = max(pos.y, 1.0f);
    pos.x = min(pos.x, lookup.w-2.0f);
    pos.y = min(pos.y, lookup.h-2.0f);

    lookup(x,y) = pos;
}

void CreateMatlabLookupTable(
        Image<float2> lookup, float fu, float fv, float u0, float v0, float k1, float k2, Mat<float,9> H_on
) {
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, lookup);
    KernCreateMatlabLookupTable<<<gridDim,blockDim>>>(lookup,fu,fv,u0,v0,k1,k2,H_on);
}

//////////////////////////////////////////////////////
// Warp image using lookup table
//////////////////////////////////////////////////////

__global__ void KernWarp(
    Image<unsigned char> out, const Image<unsigned char> in, const Image<float2> lookup
) {
    const uint x = blockIdx.x*blockDim.x + threadIdx.x;
    const uint y = blockIdx.y*blockDim.y + threadIdx.y;

    const float2 lu = lookup(x,y);
    out(x,y) = in.GetBilinear<float>(lu.x, lu.y);
}

void Warp(
    Image<unsigned char> out, const Image<unsigned char> in, const Image<float2> lookup
) {
    assert(out.w <= lookup.w && out.h <= lookup.h);
    assert(out.w <= in.w && out.h <= in.w);

    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, out);
    KernWarp<<<gridDim,blockDim>>>(out, in, lookup);

}

}
