#include "hip/hip_runtime.h"
#include "cu_dense_stereo.h"

#include "launch_utils.h"
#include "MatUtils.h"
#include "patch_score.h"
#include "disparity.h"
#include "InvalidValue.h"
#include "ImageApron.h"

namespace roo
{

const int MinDisparity = 0;
const int DefaultRad = 2;
//typedef SSNDPatchScore<float,DefaultRad,ImgAccessRaw> DefaultSafeScoreType;
typedef SANDPatchScore<float,DefaultRad,ImgAccessRaw> DefaultSafeScoreType;
//typedef SinglePixelSqPatchScore<float,ImgAccessRaw> DefaultSafeScoreType;

//////////////////////////////////////////////////////
// Cost Volume minimum
//////////////////////////////////////////////////////

template<typename Tdisp, typename Tvol>
__global__ void KernCostVolMinimum(Image<Tdisp> disp, Volume<Tvol> vol, unsigned maxDispVal)
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    Tdisp bestd = 0;
    Tvol bestc = vol(x,y,0);

    const int maxDisp = min(maxDispVal, x+1);
    for(int d=1; d < maxDisp; ++d) {
        const Tvol c = vol(x,y,d);
        if(c < bestc) {
            bestc = c;
            bestd = d;
        }
    }
    disp(x,y) = bestd;
}


template<typename Tdisp, typename Tvol>
void CostVolMinimum(Image<Tdisp> disp, Volume<Tvol> vol, unsigned maxDisp)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim,gridDim,disp);
    KernCostVolMinimum<Tdisp,Tvol><<<gridDim,blockDim>>>(disp,vol,maxDisp);
}

template void CostVolMinimum<>(Image<char>,Volume<float>,unsigned);
template void CostVolMinimum<>(Image<char>,Volume<int>,unsigned);
template void CostVolMinimum<>(Image<char>,Volume<unsigned int>,unsigned);
template void CostVolMinimum<>(Image<char>,Volume<unsigned short>,unsigned);
template void CostVolMinimum<>(Image<char>,Volume<unsigned char>,unsigned);
template void CostVolMinimum<>(Image<float>,Volume<float>,unsigned);
template void CostVolMinimum<>(Image<float>,Volume<unsigned short>,unsigned);

//////////////////////////////////////////////////////
// Cost Volume minimum subpix refinement
//////////////////////////////////////////////////////

template<typename Tdisp, typename Tvol>
__global__ void KernCostVolMinimumSubpix(Image<Tdisp> disp, Volume<Tvol> vol, unsigned maxDispVal, float sd)
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    if( x < disp.w && y < disp.h ) {
        Tdisp bestd = 0;
        Tvol bestc = 1E10;

        for(int d=0; d < maxDispVal; ++d) {
            const int xr = x + sd*d;
            if(0 <= xr && xr < vol.w) {
                const Tvol c = vol(x,y,d);
                if(c < bestc) {
                    bestc = c;
                    bestd = d;
                }
            }
        }

        Tdisp out = bestd;

        const int bestxr = x + sd*bestd;
        if( 0 < bestxr && bestxr < vol.w-1) {
            // Fit parabola to neighbours
            const float dl = bestd-1;
            const float dr = bestd+1;
            const float sl = vol(x,y,dl);
            const float sr = vol(x,y,dr);

            const float subpixdisp = bestd - (sr-sl) / (2*(sr-2*bestc+sl));

            // Minima of parabola

            // Check that minima is sensible. Otherwise assume bad data.
            if( dl < subpixdisp && subpixdisp < dr ) {
                out = subpixdisp;
            }
        }

        disp(x,y) = out;
    }
}

void CostVolMinimumSubpix(Image<float> disp, Volume<float> vol, unsigned maxDisp, float sd)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim,gridDim,disp);
    KernCostVolMinimumSubpix<float,float><<<gridDim,blockDim>>>(disp,vol,maxDisp,sd);
}

//////////////////////////////////////////////////////
// Cost Volume minimum square penalty subpix refinement
//////////////////////////////////////////////////////

template<typename Tdisp, typename Tvol>
__global__ void KernCostVolMinimumSquarePenaltySubpix(Image<Tdisp> imga, Volume<Tvol> vol, Image<float> imgd, unsigned maxDispVal, float sd, float lambda, float theta)
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    if( x < imga.w && y < imga.h ) {
        const float lastd = imgd(x,y);
        const float inv2theta = 1.0f / (2.0f*theta);

        Tdisp bestd = 0;
        Tvol bestc = inv2theta*lastd*lastd + lambda * vol(x,y,0);

        for(int d=1; d < maxDispVal; ++d) {
            const int xr = x + sd*d;
            if(0 <= xr && xr < vol.w) {
                const float ddif = lastd - d;
                const Tvol c = inv2theta*ddif*ddif + lambda * vol(x,y,d);
                if(c < bestc) {
                    bestc = c;
                    bestd = d;
                }
            }
        }

        Tdisp out = bestd;

        const int bestxr = x + sd*bestd;
        if( 0 < bestxr && bestxr < vol.w-1) {
            // Newton Step
            const float dl = bestd-1;
            const float dr = bestd+1;
            const float sl = inv2theta*(lastd-dl)*(lastd-dl) + lambda * vol(x,y,dl); //vol(x,y,d3);
            const float sr = inv2theta*(lastd-dr)*(lastd-dr) + lambda * vol(x,y,dr); //vol(x,y,d1);

            const float subpixdisp = bestd - (sr-sl) / (2*(sr-2*bestc+sl));

            // Check that minima is sensible. Otherwise assume bad data.
            if( dl < subpixdisp && subpixdisp < dr ) {
                out = subpixdisp;
            }
        }

        imga(x,y) = out;
    }
}

void CostVolMinimumSquarePenaltySubpix(Image<float> imga, Volume<float> vol, Image<float> imgd, unsigned maxDisp, float sd, float lambda, float theta)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim,gridDim,imga);
    KernCostVolMinimumSquarePenaltySubpix<float,float><<<gridDim,blockDim>>>(imga,vol,imgd,maxDisp,sd,lambda,theta);
}

//////////////////////////////////////////////////////
// Edge Weight
//////////////////////////////////////////////////////

__global__ void KernExponentialEdgeWeight(Image<float> imgw, const Image<float> imgi, float alpha, float beta)
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    if( x < imgi.w && y < imgi.h ) {
        float2 grad = make_float2(0,0);
        if(0<x && x<imgi.w-1) grad.x = imgi.GetCentralDiffDx<float>(x,y);
        if(0<y && y<imgi.h-1) grad.y = imgi.GetCentralDiffDy<float>(x,y);
//        if(0<x && x<imgi.w) grad.x = imgi.GetBackwardDiffDx<float>(x,y);
//        if(0<y && y<imgi.h) grad.y = imgi.GetBackwardDiffDy<float>(x,y);

        const float w = expf( -alpha * powf(sqrt(grad.x*grad.x + grad.y*grad.y),beta) );
        imgw(x,y) = w;
    }
}

void ExponentialEdgeWeight(Image<float> imgw, const Image<float> imgi, float alpha, float beta)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim,gridDim,imgw);
    KernExponentialEdgeWeight<<<gridDim,blockDim>>>(imgw,imgi,alpha,beta);
}


//////////////////////////////////////////////////////
// Scanline rectified dense stereo
//////////////////////////////////////////////////////

template<typename TD, typename TI, typename Score>
__global__ void KernDenseStereo(
    Image<TD> dDisp, Image<TI> dCamLeft, Image<TI> dCamRight, TD maxDispVal, TD dispStep, float acceptThresh
) {
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    TD bestDisp = InvalidValue<TD>::Value();

    if( Score::width  <= x && x < (dCamLeft.w - Score::width) &&
        Score::height <= y && y < (dCamLeft.h - Score::height) )
    {
        // Search for best matching pixel
        float bestScore = 1E+36;

        TD sndBestDisp = InvalidValue<TD>::Value();
        float sndBestScore = 1E+37;

        TD minDisp = min(maxDispVal, (TD)0);
        TD maxDisp = max((TD)0, maxDispVal);
        minDisp = max((int)minDisp, -(int)( ((int)dCamLeft.w - (int)Score::width) - (int)x));
        maxDisp = min((int)maxDisp, (int)(x + Score::width));

        for(TD c = minDisp; c <= maxDisp; c += dispStep ) {
            const float score =  Score::Score(dCamLeft, x,y, dCamRight, x-c, y);
            if(score < bestScore) {
                sndBestDisp = bestDisp;
                sndBestScore = bestScore;
                bestDisp = c;
                bestScore = score;
            }else if( score <= sndBestScore) {
                sndBestDisp = c;
                sndBestScore = score;
            }
        }
        if(abs(bestDisp-sndBestDisp) > 1) {
            const float cd = (sndBestScore - bestScore) / bestScore;
            if( cd < acceptThresh ) {
                bestDisp = InvalidValue<TD>::Value();
            }
        }
    }

    dDisp(x,y) = bestDisp;
}

const int MAXBW = 512;

//template<typename TD, typename TI, typename Score>
//__global__ void KernDenseStereo(
//    Image<TD> dDisp, Image<TI> dCamLeft, Image<TI> dCamRight, TD maxDispVal, TD dispStep, float acceptThresh
//) {
//    const int x = blockIdx.x*blockDim.x + threadIdx.x;
//    const int y = blockIdx.y*blockDim.y + threadIdx.y;

//    const int W = Score::width;
//    const int RAD = W / 2;

////    TI patch[W*W];

//    // only enough shared mem to cache right image
////    __shared__ ImageApronRows<TI,MAXBW,1,RAD> apron_l;
//    __shared__ ImageApronRows<TI,MAXBW,1,RAD> apron_r;
////    __shared__ ImageApronRows<TI,MAXBW,1,0> col_avg_l;
////    __shared__ ImageApronRows<TI,MAXBW,1,0> col_avg_r;
////    __shared__ ImageApronRows<TI,MAXBW,1,0> avg_l;
////    __shared__ ImageApronRows<TI,MAXBW,1,0> avg_r;

//    ///////////////////////////////////
////    // Cache line of right/left image +/- RAD
////    apron_l.CacheImage(dCamLeft);
//    apron_r.CacheImage(dCamRight);

////    __syncthreads();

////    ///////////////////////////////////
////    // Cache sum of colums for norm
//////    int colsuml = 0;
////    int colsumr = 0;
////#pragma unroll
////    for(int i=-RAD; i<=RAD; ++i) {
//////        colsuml += apron_l.GetRelThread(0,i);
////        colsumr += apron_r.GetRelThread(0,i);
////    }
//////    col_avg_l.GetRelThread(0,0) = colsuml / W;
////    col_avg_r.GetRelThread(0,0) = colsumr / W;
////    __syncthreads();

////    ///////////////////////////////////
////    // Cache sum of block for norm
//////    int suml = 0;
////    int sumr = 0;
////#pragma unroll
////    for(int i=-RAD; i<=RAD; ++i) {
//////        suml += col_avg_l.GetRelThreadClampX(i,0);
////        sumr += col_avg_r.GetRelThreadClampX(i,0);
////    }
//////    avg_l.GetRelThread(0,0) = suml / W;
////    avg_r.GetRelThread(0,0) = sumr / W;

//    ///////////////////////////////////
//    // Cache left patch, compute mean
//////    int sum_l = 0;
////    for(int r=-RAD; r<= RAD; ++r) {
////#pragma unroll
////        for(int c=-RAD; c<=RAD; ++c) {
////            const TI val = dCamLeft.GetWithClampedRange(x+c, y+r);
////            patch[(RAD+r)*W+(RAD+c)] = val;
//////            sum_l += val;
////        }
////    }
////    const TI avg_l = sum_l / (W*W);

//    __syncthreads();

//    TD bestDisp = InvalidValue<TD>::Value();

//    if( maxDispVal+Score::width <= x && x < (dCamLeft.w - Score::width) &&
//        Score::height <= y && y < (dCamLeft.h - Score::height) )
//    {
//        // Search for best matching pixel
//        float bestScore = 1E+36;

////        TD sndBestDisp = InvalidValue<TD>::Value();
////        float sndBestScore = 1E+37;

////        TD minDisp = min(maxDispVal, (TD)0);
////        TD maxDisp = max((TD)0, maxDispVal);
////        minDisp = max((int)minDisp, -(int)( ((int)dCamLeft.w - (int)Score::width) - (int)x));
////        maxDisp = min((int)maxDisp, (int)(x + Score::width));

//        for(TD c = 0; c <= maxDispVal; c += 1 ) {
//            float score = 0;

//            for(int ky=-RAD; ky <= RAD; ++ky ) {
//#pragma unroll
//                for(int kx=-RAD; kx <= RAD; ++kx ) {
////                    const int pl = apron_l.GetRelThread(kx,ky);
//                    const int pl = 0;//patch[(RAD+ky)*W+(RAD+kx)];
//                    const int pr = apron_r.GetRelThread(kx-c,ky);
//                    score += abs(pl - pr);
//                }
//            }

//////            Score::Score(dCamLeft, x,y, dCamRight, x-c, y);
//            if(score < bestScore) {
////                sndBestDisp = bestDisp;
////                sndBestScore = bestScore;
//                bestDisp = c;
//                bestScore = score;
////            }else if( score <= sndBestScore) {
////                sndBestDisp = c;
////                sndBestScore = score;
//            }
//        }
////        if(abs(bestDisp-sndBestDisp) > 1) {
////            const float cd = (sndBestScore - bestScore) / bestScore;
////            if( cd < acceptThresh ) {
////                bestDisp = InvalidValue<TD>::Value();
////            }
////        }
//    }

//    dDisp(x,y) = bestDisp;
//}

template<typename TDisp, typename TImg>
void DenseStereo(
    Image<TDisp> dDisp, const Image<TImg> dCamLeft, const Image<TImg> dCamRight,
    TDisp maxDisp, float acceptThresh, int score_rad
) {
    dim3 blockDim(dDisp.w, 1);
    dim3 gridDim(1, dDisp.h);
//    InitDimFromOutputImageOver(blockDim,gridDim,dDisp);

    const TDisp dispStep = 1;
    if( score_rad == 0 ) {
        KernDenseStereo<TDisp, TImg, SinglePixelSqPatchScore<float,ImgAccessRaw > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
    }else if(score_rad == 1 ) {
        KernDenseStereo<TDisp, TImg, SANDPatchScore<float,1,ImgAccessRaw > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
    }else if( score_rad == 2 ) {
        KernDenseStereo<TDisp, TImg, SANDPatchScore<float,2,ImgAccessRaw > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
    }else if(score_rad == 3 ) {
        KernDenseStereo<TDisp, TImg, SANDPatchScore<float,3,ImgAccessRaw > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
    }else if( score_rad == 4 ) {
        KernDenseStereo<TDisp, TImg, SANDPatchScore<float,4,ImgAccessRaw > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
    }else if(score_rad == 5 ) {
        KernDenseStereo<TDisp, TImg, SANDPatchScore<float,5,ImgAccessRaw > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
    }else if(score_rad == 6 ) {
        KernDenseStereo<TDisp, TImg, SANDPatchScore<float,6,ImgAccessRaw > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
    }else if(score_rad == 7 ) {
        KernDenseStereo<TDisp, TImg, SANDPatchScore<float,7,ImgAccessRaw > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
    }
}

template void DenseStereo<unsigned char, unsigned char>(Image<unsigned char>, const Image<unsigned char>, const Image<unsigned char>, unsigned char, float, int);
template void DenseStereo<char, unsigned char>(Image<char>, const Image<unsigned char>, const Image<unsigned char>, char, float, int);

void DenseStereoSubpix(
    Image<float> dDisp, const Image<unsigned char> dCamLeft, const Image<unsigned char> dCamRight, float maxDisp, float dispStep, float acceptThresh, int score_rad, bool score_normed
) {
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, dDisp);

//    if(score_normed) {
//        if( score_rad == 0 ) {
//            KernDenseStereo<float, unsigned char, SinglePixelSqPatchScore<float,ImgAccessBilinear<float> > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
//        }else if(score_rad == 1 ) {
//            KernDenseStereo<float, unsigned char, SANDPatchScore<float,1,ImgAccessBilinear<float> > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
//        }else if( score_rad == 2 ) {
//            KernDenseStereo<float, unsigned char, SANDPatchScore<float,2,ImgAccessBilinear<float> > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
//        }else if(score_rad == 3 ) {
//            KernDenseStereo<float, unsigned char, SANDPatchScore<float,3,ImgAccessBilinear<float> > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
//        }else if( score_rad == 4 ) {
//            KernDenseStereo<float, unsigned char, SANDPatchScore<float,4,ImgAccessBilinear<float> > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
//        }else if(score_rad == 5 ) {
//            KernDenseStereo<float, unsigned char, SANDPatchScore<float,5,ImgAccessBilinear<float> > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
//        }else if(score_rad == 6 ) {
//            KernDenseStereo<float, unsigned char, SANDPatchScore<float,6,ImgAccessBilinear<float> > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
//        }else if(score_rad == 7 ) {
//            KernDenseStereo<float, unsigned char, SANDPatchScore<float,7,ImgAccessBilinear<float> > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
//        }
//    }else{
//        if( score_rad == 0 ) {
//            KernDenseStereo<float, unsigned char, SinglePixelSqPatchScore<float,ImgAccessBilinear<float> > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
//        }else if(score_rad == 1 ) {
//            KernDenseStereo<float, unsigned char, SADPatchScore<float,1,ImgAccessBilinear<float> > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
//        }else if( score_rad == 2 ) {
//            KernDenseStereo<float, unsigned char, SADPatchScore<float,2,ImgAccessBilinear<float> > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
//        }else if(score_rad == 3 ) {
//            KernDenseStereo<float, unsigned char, SADPatchScore<float,3,ImgAccessBilinear<float> > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
//        }else if( score_rad == 4 ) {
//            KernDenseStereo<float, unsigned char, SADPatchScore<float,4,ImgAccessBilinear<float> > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
//        }else if(score_rad == 5 ) {
//            KernDenseStereo<float, unsigned char, SADPatchScore<float,5,ImgAccessBilinear<float> > ><<<gridDim,blockDim>>>(dDisp, dCamLeft, dCamRight, maxDisp, dispStep, acceptThresh);
//        }
//    }
}

//////////////////////////////////////////////////////
//////////////////////////////////////////////////////

const int RAD = 3;
const int W = 2*RAD+1;

__global__ void KernDenseStereoTest(
    Image<float> dDisp, Image<unsigned char> dCamLeft, Image<unsigned char> dCamRight, int maxDisp
) {
    const int x = threadIdx.x;
    const int y = blockIdx.y;

    __shared__ unsigned char cache_l[W][MAXBW];
    __shared__ unsigned char cache_r[W][MAXBW+1];

#pragma unroll
    for(int r=0; r<W; ++r ) {
        cache_l[r][x] = dCamLeft.Get(x,y+r-RAD);
        cache_r[r][x] = dCamRight.Get(x,y+r-RAD);
    }

    __syncthreads();

    int bestScore = 0xFFFFF;
    int bestDisp = 0;

    const int maxClipDisp = min(x-RAD,maxDisp);
    for(int d=0; d<maxClipDisp; ++d)
    {
        const int xd = x-d;
        int score = 0;
#pragma unroll
        for(int r=0; r<W; ++r) {
            score += abs(cache_l[r][x] - cache_r[r][xd]);
//            const int yr = y-RAD+r;
//            score += abs(dCamLeft(x,yr) - dCamRight(xd,yr));
        }

        if(score < bestScore) {
            bestScore = score;
            bestDisp = d;
        }
    }

    dDisp(x,y) = bestDisp;
}

void DenseStereoTest(
    Image<float> dDisp, Image<unsigned char> dCamLeft, Image<unsigned char> dCamRight, int maxDisp
) {
    const int w = dDisp.w;
    const int h = dDisp.h - 2*RAD;
    const int x = 0;
    const int y = RAD;

    dim3 blockDim(w, 1);
    dim3 gridDim(1, h);
    KernDenseStereoTest<<<gridDim,blockDim>>>(dDisp.SubImage(x,y,w,h), dCamLeft.SubImage(x,y,w,h), dCamRight.SubImage(x,y,w,h), maxDisp);
}

//////////////////////////////////////////////////////
// Check Left and Right disparity images match
//////////////////////////////////////////////////////

template<typename TD>
__global__ void KernLeftRightCheck(
    Image<TD> dispL, Image<TD> dispR, float sd, float maxDiff
) {
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    if( dispL.InBounds(x,y) ) {
        const TD dl = dispL(x,y);
        const TD xr = x + sd*dl;

        if( 0 <= xr && xr < dispR.w) {
            const TD dr = dispR(xr, y);
            if(!InvalidValue<TD>::IsValid(dr) || abs(dl - dr) > maxDiff) {
                dispL(x,y) = InvalidValue<TD>::Value();
            }
        }else{
            dispL(x,y) = InvalidValue<TD>::Value();
        }
    }
}

void LeftRightCheck(Image<char> dispL, Image<char> dispR, int sd, int maxDiff)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim,gridDim, dispL);
    KernLeftRightCheck<char><<<gridDim,blockDim>>>(dispL, dispR, sd, maxDiff);
}

void LeftRightCheck(Image<float> dispL, Image<float> dispR, float sd, float maxDiff)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim,gridDim, dispL);
    KernLeftRightCheck<float><<<gridDim,blockDim>>>(dispL, dispR, sd, maxDiff);
}

//////////////////////////////////////////////////////
// Visualise cross section of disparity image
//////////////////////////////////////////////////////

template<typename TD, typename TI, typename Score>
__global__ void KernDisparityImageCrossSection(
    Image<TD> dScore, Image<unsigned char> dDisp, Image<TI> dCamLeft, Image<TI> dCamRight, int y
) {
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int c = blockIdx.y*blockDim.y + threadIdx.y;

    const int rx = x-c;
    const float score = ( 0<= rx && rx < dCamRight.w ) ? Score::Score(dCamLeft, x,y, dCamRight, rx, y) : 0;

    const unsigned char mindisp = dDisp(x,y);
    const float show = sqrt(score / Score::area) / 255.0f;

    dScore(x,c) = show * make_float4( 1,1,1,1);
}

void DisparityImageCrossSection(
    Image<float4> dScore, Image<unsigned char> dDisp, const Image<unsigned char> dCamLeft, const Image<unsigned char> dCamRight, int y
) {
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, dScore);
    KernDisparityImageCrossSection<float4, unsigned char, DefaultSafeScoreType><<<gridDim,blockDim>>>(dScore, dDisp, dCamLeft, dCamRight, y);
}

//////////////////////////////////////////////////////
// Scanline rectified dense stereo sub-pixel refinement
//////////////////////////////////////////////////////

template<typename TDo, typename TDi, typename TI, typename Score>
__global__ void KernDenseStereoSubpixelRefine(
    Image<TDo> dDispOut, const Image<TDi> dDisp, const Image<TI> dCamLeft, const Image<TI> dCamRight
) {
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    const int bestDisp = dDisp(x,y);

    // Ignore things at infinity
    if(bestDisp < MinDisparity) {
        dDispOut(x,y) = InvalidValue<TDo>::Value();
        return;
    }

    // Fit parabola to neighbours
    const float d1 = bestDisp+1;
    const float d2 = bestDisp;
    const float d3 = bestDisp-1;
    const float s1 = Score::Score(dCamLeft, x,y, dCamRight, x-d1,y);
    const float s2 = Score::Score(dCamLeft, x,y, dCamRight, x-d2,y);
    const float s3 = Score::Score(dCamLeft, x,y, dCamRight, x-d3,y);

    // Cooefficients of parabola through (d1,s1),(d2,s2),(d3,s3)
    const float denom = (d1 - d2)*(d1 - d3)*(d2 - d3);
    const float A = (d3 * (s2 - s1) + d2 * (s1 - s3) + d1 * (s3 - s2)) / denom;
    const float B = (d3*d3 * (s1 - s2) + d2*d2 * (s3 - s1) + d1*d1 * (s2 - s3)) / denom;
//    const float C = (x2 * x3 * (x2 - x3) * y1 + x3 * x1 * (x3 - x1) * y2 + x1 * x2 * (x1 - x2) * y3) / denom;

    // Minima of parabola
    const float newDisp = -B / (2*A);

    // Check that minima is sensible. Otherwise assume bad data.
    if( d3 < newDisp && newDisp < d1 ) {
        dDispOut(x,y) = newDisp;
    }else{
//        dDisp(x,y) = bestDisp / maxDisp;
        dDispOut(x,y) = InvalidValue<TDo>::Value();
    }
}

void DenseStereoSubpixelRefine(
    Image<float> dDispOut, const Image<unsigned char> dDisp, const Image<unsigned char> dCamLeft, const Image<unsigned char> dCamRight
) {
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, dDisp);
    KernDenseStereoSubpixelRefine<float,unsigned char,unsigned char, DefaultSafeScoreType><<<gridDim,blockDim>>>(dDispOut, dDisp, dCamLeft, dCamRight);
}

//////////////////////////////////////////////////////
// Upgrade disparity image to vertex array
//////////////////////////////////////////////////////

__global__ void KernDisparityImageToVbo(
    Image<float4> dVbo, const Image<float> dDisp, float baseline, float fu, float fv, float u0, float v0
) {
    const int u = blockIdx.x*blockDim.x + threadIdx.x;
    const int v = blockIdx.y*blockDim.y + threadIdx.y;
    dVbo(u,v) = DepthFromDisparity(u,v, dDisp(u,v), baseline, fu, fv, u0, v0, MinDisparity);
}

void DisparityImageToVbo(Image<float4> dVbo, const Image<float> dDisp, float baseline, float fu, float fv, float u0, float v0)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, dVbo);
    KernDisparityImageToVbo<<<gridDim,blockDim>>>(dVbo, dDisp, baseline, fu, fv, u0, v0);
}

//////////////////////////////////////////////////////
// Cost Volume
//////////////////////////////////////////////////////

void CostVolumeZero(Volume<CostVolElem> costvol )
{
    CostVolElem initial;
    initial.sum = 0;
    initial.n = 0;
    costvol.Fill(initial);
}

//////////////////////////////////////////////////////

template<typename TD, typename TI, typename Score>
__global__ void KernCostVolumeFromStereo(
    Volume<CostVolElem> dvol, Image<TI> dimgl, Image<TI> dimgr
) {
    const int u = blockIdx.x*blockDim.x + threadIdx.x;
    const int v = blockIdx.y*blockDim.y + threadIdx.y;
    const int d = blockIdx.z*blockDim.z + threadIdx.z;

    if( u-d >= (int)Score::rad) {
        CostVolElem elem;
        elem.sum = Score::Score(dimgl, u,v, dimgr, u-d, v) / Score::area;
        elem.n = 1;
        dvol(u,v,d) = elem;
    }
}

void CostVolumeFromStereo(Volume<CostVolElem> dvol, Image<unsigned char> dimgl, Image<unsigned char> dimgr )
{
    dim3 blockDim(8,8,8);
    dim3 gridDim(dvol.w / blockDim.x, dvol.h / blockDim.y, dvol.d / blockDim.z);
    KernCostVolumeFromStereo<unsigned char, unsigned char, DefaultSafeScoreType><<<gridDim,blockDim>>>(dvol,dimgl,dimgr);
}

//////////////////////////////////////////////////////

template<typename TI, typename Score>
__global__ void KernAddToCostVolume(
    Volume<CostVolElem> dvol, const Image<TI> dimgv,
    const Image<TI> dimgc, Mat<float,3,4> KT_cv,
    float fu, float fv, float u0, float v0,
    float baseline
){
    const int u = blockIdx.x*blockDim.x + threadIdx.x;
    const int v = blockIdx.y*blockDim.y + threadIdx.y;
    const int d = blockIdx.z*blockDim.z + threadIdx.z;

    float3 Pv;
    Pv.z = fu * baseline / d;
    Pv.x = Pv.z * (u-u0) / fu;
    Pv.y = Pv.z * (v-v0) / fv;

    const float3 KPc = KT_cv * Pv;
    const float2 pc = dn(KPc);

    if( KPc.z > 0 && dimgc.InBounds(pc.x, pc.y,5) ) {
//        vol(u,v,d) = 1.0f;
        const float score =  Score::Score(dimgv, u,v, dimgc, pc.x, pc.y) / (float)(Score::area);
//        const float score = (dimgv(u,v) - dimgc.template GetBilinear<float>(pc)) / 255.0f;
        CostVolElem elem = dvol(u,v,d);
        elem.sum += score;
        elem.n += 1;
        dvol(u,v,d) = elem;
    }
}

void CostVolumeAdd(Volume<CostVolElem> dvol, const Image<unsigned char> dimgv,
    const Image<unsigned char> dimgc, Mat<float,3,4> KT_cv,
    float fu, float fv, float u0, float v0,
    float baseline, int levels
) {
    dim3 blockDim(8,8,8);
    dim3 gridDim(dvol.w / blockDim.x, dvol.h / blockDim.y, dvol.d / blockDim.z);
    KernAddToCostVolume<unsigned char, SANDPatchScore<float,DefaultRad,ImgAccessBilinearClamped<float> > ><<<gridDim,blockDim>>>(dvol,dimgv,dimgc, KT_cv, fu,fv,u0,v0, baseline);
}

//////////////////////////////////////////////////////

template<typename Tdisp>
__global__ void KernCostVolMinimum(Image<Tdisp> disp, Volume<CostVolElem> vol)
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    Tdisp bestd = 0;
    float bestc = 1E30;

    unsigned maxDisp = vol.d;
#pragma unroll
    for(int d=0; d < maxDisp; ++d) {
        const CostVolElem elem = vol(x,y,d);
        const float c = (elem.sum / elem.n);
        if(c < bestc) {
            bestc = c;
            bestd = d;
        }
    }
    disp(x,y) = bestd;
}


void CostVolMinimum(Image<float> disp, Volume<CostVolElem> vol)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim,gridDim,disp);
    KernCostVolMinimum<float><<<gridDim,blockDim>>>(disp,vol);
}

//////////////////////////////////////////////////////

__global__ void KernCostVolumeCrossSection(
    Image<float> dScore, Image<CostVolElem> dCostVolSlice
) {
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int d = blockIdx.y*blockDim.y + threadIdx.y;

    if( dCostVolSlice.InBounds(x,d) )
    {
        CostVolElem elem = dCostVolSlice(x,d);
        const float score = (elem.sum / elem.n) / 255.0f;
        dScore(x,d) = score;
    }else{
        dScore(x,d) = 0.0f / 0.0f;
    }
}

void CostVolumeCrossSection(
    Image<float> dScore, Volume<CostVolElem> dCostVol, int y
) {
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, dScore);
    KernCostVolumeCrossSection<<<gridDim,blockDim>>>(dScore, dCostVol.ImageXZ(y));
}

//////////////////////////////////////////////////////

template<typename To, typename Ti>
__global__ void KernFilterDispGrad(Image<To> dOut, Image<Ti> dIn, float threshold )
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    const float dx = dOut.template GetCentralDiffDx<float>(x,y);
    const float dy = dOut.template GetCentralDiffDy<float>(x,y);
    const bool valid = dx*dx + dy*dy < threshold;

    dOut(x,y) = valid ? dIn(x,y) : -1;
}

void FilterDispGrad(
    Image<float> dOut, Image<float> dIn, float threshold
) {
    dim3 blockDim, gridDim;
    InitDimFromOutputImage(blockDim,gridDim, dOut, 16, 16);
    KernFilterDispGrad<float,float><<<gridDim,blockDim>>>(dOut, dIn, threshold);
}

//////////////////////////////////////////////////////
// Cost volume with truncated grad and abs. diff. score
// Fast Cost-Volume Filtering for Visual Correspondence and Beyond
// Christoph Rhemann, Asmaa Hosni, Michael Bleyer, Carsten Rother, Margrit Gelautz
//////////////////////////////////////////////////////

template<typename Tout, typename Tin>
__global__ void KernCostVolumeFromStereoTruncatedAbsAndGrad(
    Volume<Tout> dvol, Image<Tin> dimgl, Image<Tin> dimgr, float sd,
    float alpha, float r1, float r2
) {
    const int u = blockIdx.x*blockDim.x + threadIdx.x;
    const int v = blockIdx.y*blockDim.y + threadIdx.y;
    const int d = blockIdx.z*blockDim.z + threadIdx.z;

    const int r = u + sd*d;
    if( 0 <= r && r < dimgr.w ) {
        const float absI = abs( (float)dimgr(r,v) - (float)dimgl(u,v));
        const float absGrad = abs( dimgr.template GetCentralDiffDx<float>(r,v) - dimgl.template GetCentralDiffDx<float>(u,v) );
        const Tout cost = (1.0f-alpha)*min(absI,r1) + alpha*min(absGrad,r2);
        dvol(u,v,d) = cost;
    }else{
        dvol(u,v,d) = (1.0f-alpha)*r1 + alpha*r2;
    }
}

void CostVolumeFromStereoTruncatedAbsAndGrad(Volume<float> dvol, Image<float> dimgl, Image<float> dimgr, float sd, float alpha, float r1, float r2 )
{
    dim3 blockDim(8,8,8);
    dim3 gridDim( ceil(dvol.w / (float)blockDim.x), ceil(dvol.h / (float)blockDim.y), ceil(dvol.d / (float)blockDim.z) );
    KernCostVolumeFromStereoTruncatedAbsAndGrad<float,float><<<gridDim,blockDim>>>(dvol,dimgl,dimgr,sd, alpha,r1,r2);
}



}
