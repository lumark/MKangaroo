#include "hip/hip_runtime.h"
#include "cu_census.h"

#include "MatUtils.h"
#include "hamming_distance.h"
#include "launch_utils.h"
#include "InvalidValue.h"
#include "CUDA_SDK/sharedmem.h"

namespace roo
{

const int MaxImageWidth = 1024;

//////////////////////////////////////////////////////
// Census transform, 9x7 window
//////////////////////////////////////////////////////

template<typename Tout, typename Tin>
__global__ void KernCensus9x7(Image<Tout> census, Image<Tin> img)
{
    const int WRAD = 4;
    const int HRAD = 3;

    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    if( img.InBounds(x,y) ) {
        const Tin p = img(x,y);

        Tout out = 0;
        Tout bit = 1;

        for(int r=-HRAD; r <= HRAD; ++r) {
#pragma unroll
            for(int c=-WRAD; c <= WRAD; ++c ) {
                const Tin q = img.GetWithClampedRange(x+c,y+r);
                if( q < p ) {
                    out |= bit;
                }
                bit <<= 1;
            }
        }

        census(x,y) = out;
    }
}

//////////////////////////////////////////////////////
// Census transform, 11x11 window
//////////////////////////////////////////////////////

template<typename Tin>
__global__ void KernCensus11x11(Image<ulong2> census, Image<Tin> img)
{
    const int WRAD = 5;
    const int HRAD = 5;

    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    if( img.InBounds(x,y) ) {
        const Tin p = img(x,y);

        ulong2 out = make_ulong2(0,0);
        unsigned long bit = 1;

        for(int r=-HRAD; r < 0; ++r) {
#pragma unroll
            for(int c=-WRAD; c <= WRAD; ++c ) {
                const Tin q = img.GetWithClampedRange(x+c,y+r);
                if( q < p ) {
                    out.x |= bit;
                }
                bit <<= 1;
            }
        }

#pragma unroll
        for(int c=-WRAD; c <= 0; ++c ) {
            const Tin q = img.GetWithClampedRange(x+c,y);
            if( q < p ) {
                out.x |= bit;
            }
            bit <<= 1;
        }

        bit = 1;
#pragma unroll
        for(int c=1; c <= WRAD; ++c ) {
            const Tin q = img.GetWithClampedRange(x+c,y);
            if( q < p ) {
                out.y |= bit;
            }
            bit <<= 1;
        }

        for(int r=1; r <= HRAD; ++r) {
#pragma unroll
            for(int c=-WRAD; c <= WRAD; ++c ) {
                const Tin q = img.GetWithClampedRange(x+c,y+r);
                if( q < p ) {
                    out.y |= bit;
                }
                bit <<= 1;
            }
        }

        census(x,y) = out;
    }
}

//////////////////////////////////////////////////////
// Census transform, 16x16 window
//////////////////////////////////////////////////////

template<typename Tin>
__global__ void KernCensus16x16(Image<ulong4> census, Image<Tin> img)
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

    if( img.InBounds(x,y) ) {
        const Tin p = img(x,y);

        ulong4 out = make_ulong4(0,0,0,0);
        unsigned long bit = 1;

        for(int r=-8; r < -4; ++r) {
#pragma unroll
            for(int c=-4; c < 4; ++c ) {
                const Tin q = img.GetWithClampedRange(x+c,y+r);
                if( q < p ) {
                    out.x |= bit;
                }
                bit <<= 1;
            }
        }

        bit = 1;
        for(int r=-4; r < 0; ++r) {
#pragma unroll
            for(int c=-4; c < 4; ++c ) {
                const Tin q = img.GetWithClampedRange(x+c,y+r);
                if( q < p ) {
                    out.y |= bit;
                }
                bit <<= 1;
            }
        }

        bit = 1;
        for(int r=0; r < 4; ++r) {
#pragma unroll
            for(int c=-4; c < 4; ++c ) {
                const Tin q = img.GetWithClampedRange(x+c,y+r);
                if( q < p ) {
                    out.z |= bit;
                }
                bit <<= 1;
            }
        }

        bit = 1;
        for(int r=4; r < 8; ++r) {
#pragma unroll
            for(int c=-4; c < 4; ++c ) {
                const Tin q = img.GetWithClampedRange(x+c,y+r);
                if( q < p ) {
                    out.w |= bit;
                }
                bit <<= 1;
            }
        }
        census(x,y) = out;
    }

}


void Census(Image<unsigned long> census, Image<unsigned char> img)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim,gridDim,img);
    KernCensus9x7<unsigned long, unsigned char><<<gridDim,blockDim>>>(census,img);
}

void Census(Image<ulong2> census, Image<unsigned char> img)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim,gridDim,img);
    KernCensus11x11<unsigned char><<<gridDim,blockDim>>>(census,img);
}

void Census(Image<ulong4> census, Image<unsigned char> img)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim,gridDim,img);
    KernCensus16x16<unsigned char><<<gridDim,blockDim>>>(census,img);
}

void Census(Image<unsigned long> census, Image<float> img)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim,gridDim,img);
    KernCensus9x7<unsigned long, float><<<gridDim,blockDim>>>(census,img);
}

void Census(Image<ulong2> census, Image<float> img)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim,gridDim,img);
    KernCensus11x11<float><<<gridDim,blockDim>>>(census,img);
}

void Census(Image<ulong4> census, Image<float> img)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim,gridDim,img);
    KernCensus16x16<float><<<gridDim,blockDim>>>(census,img);
}

//////////////////////////////////////////////////////
// Census Stereo
//////////////////////////////////////////////////////

template<typename T>
__global__ void KernCensusStereo(Image<char> disp, Image<T> left, Image<T> right, int maxDispVal)
{
    const int x = threadIdx.x;
    const int y = blockIdx.y;

    __shared__ T cache_r[MaxImageWidth];
    cache_r[x] = right(x,y);
    __syncthreads();

    const T p = left(x,y);

    unsigned bestScore = 0xFFFFF;
    int bestDisp = InvalidValue<char>::Value();

    int minDisp = min(maxDispVal, 0);
    int maxDisp = max(0, maxDispVal);
    minDisp = max(minDisp, x - ((int)left.w-1));
    maxDisp = min(maxDisp, x);

    for(int d=minDisp; d< maxDisp; ++d)
    {
        const int xd = x-d;
        const T q = cache_r[xd]; //right(xd,y);
        const unsigned score = HammingDistance(p,q);

        if(score < bestScore) {
            bestScore = score;
            bestDisp = d;
        }
    }

    disp(x,y) = bestDisp;
}

void CensusStereo(Image<char> disp, Image<unsigned long> left, Image<unsigned long> right, int maxDisp)
{
    dim3 blockDim(disp.w, 1);
    dim3 gridDim(1, disp.h);
    KernCensusStereo<unsigned long><<<gridDim,blockDim>>>(disp,left,right,maxDisp);
}

//////////////////////////////////////////////////////
// Build Census Cost volume
//////////////////////////////////////////////////////

template<typename Tvol, typename T>
__global__ void KernCensusStereoVolume(Volume<Tvol> vol, Image<T> left, Image<T> right, int maxDispVal, float sd)
{
    const int x = threadIdx.x;
    const int y = blockIdx.y;

    SharedMemory<T> shared;
    T* cache_r = shared.getPointer();
    cache_r[x] = right(x,y);
    __syncthreads();

    const T p = left(x,y);

//    const int maxDisp = min(maxDispVal, x+1);

    for(int d=0; d< maxDispVal; ++d)
    {
        const int xd = x + sd*d;
        Tvol score;
        if(0 <= xd && xd < right.w) {
            const T q = cache_r[xd]; //right(xd,y);
            score = HammingDistance(p,q) / (float)(sizeof(T)*8);
        }else{
            score = 0.5;
        }
        vol(x,y,d) = score;
    }
}

template<typename Tvol, typename T>
void CensusStereoVolume(Volume<Tvol> vol, Image<T> left, Image<T> right, int maxDisp, float sd)
{
    dim3 blockDim(left.w, 1);
    dim3 gridDim(1, left.h);
    KernCensusStereoVolume<Tvol,T><<<gridDim,blockDim,sizeof(T)*left.w>>>(vol,left,right,maxDisp, sd);
}

template KANGAROO_EXPORT void CensusStereoVolume(Volume<unsigned short> vol, Image<unsigned long> left, Image<unsigned long> right, int maxDisp, float);
template KANGAROO_EXPORT void CensusStereoVolume(Volume<unsigned short> vol, Image<ulong2> left, Image<ulong2> right, int maxDisp, float);
template KANGAROO_EXPORT void CensusStereoVolume(Volume<unsigned short> vol, Image<ulong4> left, Image<ulong4> right, int maxDisp, float);
template KANGAROO_EXPORT void CensusStereoVolume(Volume<float> vol, Image<unsigned long> left, Image<unsigned long> right, int maxDisp, float);
template KANGAROO_EXPORT void CensusStereoVolume(Volume<float> vol, Image<ulong2> left, Image<ulong2> right, int maxDisp, float);
template KANGAROO_EXPORT void CensusStereoVolume(Volume<float> vol, Image<ulong4> left, Image<ulong4> right, int maxDisp, float);

}
