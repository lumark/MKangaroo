#include "hip/hip_runtime.h"
#include "cu_raycast.h"

#include "MatUtils.h"
#include "launch_utils.h"

namespace roo
{

//////////////////////////////////////////////////////
// Phong shading.
//////////////////////////////////////////////////////

__host__ __device__ inline
float PhongShade(const float3 p_c, const float3 n_c)
{
    const float ambient = 0.4;
    const float diffuse = 0.4;
    const float specular = 0.2;
    const float3 eyedir = -1.0f * p_c / length(p_c);
    const float3 _lightdir = make_float3(0.4,0.4,-1);
    const float3 lightdir = _lightdir / length(_lightdir);
    const float ldotn = dot(lightdir,n_c);
    const float3 lightreflect = 2*ldotn*n_c + (-1.0) * lightdir;
    const float edotr = fmaxf(0,dot(eyedir,lightreflect));
    const float spec = edotr*edotr*edotr*edotr*edotr*edotr*edotr*edotr*edotr*edotr;
    return ambient + diffuse * ldotn  + specular * spec;
}

//////////////////////////////////////////////////////
// Raycast SDF
//////////////////////////////////////////////////////

__global__ void KernRaycastSdf(Image<float> imgdepth, Image<float4> norm, Image<float> img, const BoundedVolume<SDF_t> vol, const Mat<float,3,4> T_wc, ImageIntrinsics K, float near, float far, float trunc_dist, bool subpix )
{
    const int u = blockIdx.x*blockDim.x + threadIdx.x;
    const int v = blockIdx.y*blockDim.y + threadIdx.y;

    if( u < img.w && v < img.h ) {
        const float3 c_w = SE3Translation(T_wc);
        const float3 ray_c = K.Unproject(u,v);
        const float3 ray_w = mulSO3(T_wc, ray_c);

        // Raycast bounding box to find valid ray segment of sdf
        // http://www.cs.utah.edu/~awilliam/box/box.pdf
        const float3 tminbound = (vol.bbox.Min() - c_w) / ray_w;
        const float3 tmaxbound = (vol.bbox.Max() - c_w) / ray_w;
        const float3 tmin = fminf(tminbound,tmaxbound);
        const float3 tmax = fmaxf(tminbound,tmaxbound);
        const float max_tmin = fmaxf(fmaxf(fmaxf(tmin.x, tmin.y), tmin.z), near);
        const float min_tmax = fminf(fminf(fminf(tmax.x, tmax.y), tmax.z), far);

        float depth = 0.0f;

        // If ray intersects bounding box
        if(max_tmin < min_tmax ) {
            // Go between max_tmin and min_tmax
            float lambda = max_tmin;
            float last_sdf = 0.0f/0.0f;
            float min_delta_lambda = vol.VoxelSizeUnits().x;
            float delta_lambda = 0;

            // March through space
            while(lambda < min_tmax) {
                const float3 pos_w = c_w + lambda * ray_w;
                const float sdf = vol.GetUnitsTrilinearClamped(pos_w);

                if( sdf <= 0 ) {
                    if( last_sdf > 0) {
                        // surface!
                        if(subpix) {
                            lambda = lambda + delta_lambda * sdf / (last_sdf - sdf);
                        }
                        depth = lambda;
                    }
                    break;
                }
                delta_lambda = sdf > 0 ? fmaxf(sdf, min_delta_lambda) : trunc_dist;
                lambda += delta_lambda;
                last_sdf = sdf;
            }
        }

        // Compute normal
        const float3 pos_w = c_w + depth * ray_w;
        const float3 _n_w = vol.GetUnitsBackwardDiffDxDyDz(pos_w);
        const float len_n_w = length(_n_w);
        const float3 n_w = len_n_w > 0 ? _n_w / len_n_w : make_float3(0,0,1);
        const float3 n_c = mulSO3inv(T_wc,n_w);
        const float3 p_c = depth * ray_c;

        if(depth > 0 ) {
//          img(u,v) = (depth - near) / (far - near);
            imgdepth(u,v) = depth;
            img(u,v) = PhongShade(p_c, n_c);
//            norm(u,v) = make_float4(0.5,0.5,0.5,1) + make_float4(n_c, 0) /2.0f;
            norm(u,v) = make_float4(n_c, 1);
        }else{
            imgdepth(u,v) = 0.0f/0.0f;
            img(u,v) = 0;
            norm(u,v) = make_float4(0,0,0,0);
        }
    }
}

void RaycastSdf(Image<float> depth, Image<float4> norm, Image<float> img, const BoundedVolume<SDF_t> vol, const Mat<float,3,4> T_wc, ImageIntrinsics K, float near, float far, float trunc_dist, bool subpix )
{
    dim3 blockDim, gridDim;
//    InitDimFromOutputImageOver(blockDim, gridDim, img, 16, 16);
    InitDimFromOutputImageOver(blockDim, gridDim, img);
    KernRaycastSdf<<<gridDim,blockDim>>>(depth, norm, img, vol, T_wc, K, near, far, trunc_dist, subpix);
    GpuCheckErrors();
}

//////////////////////////////////////////////////////
// Raycast Color SDF
//////////////////////////////////////////////////////

__global__ void KernRaycastSdf(Image<float> imgdepth, Image<float4> norm, Image<float> img, const BoundedVolume<SDF_t> vol, const BoundedVolume<float> colorVol, const Mat<float,3,4> T_wc, ImageIntrinsics K, float near, float far, float trunc_dist, bool subpix )
{
    const int u = blockIdx.x*blockDim.x + threadIdx.x;
    const int v = blockIdx.y*blockDim.y + threadIdx.y;

    if( u < img.w && v < img.h ) {
        const float3 c_w = SE3Translation(T_wc);
        const float3 ray_c = K.Unproject(u,v);
        const float3 ray_w = mulSO3(T_wc, ray_c);

        // Raycast bounding box to find valid ray segment of sdf
        // http://www.cs.utah.edu/~awilliam/box/box.pdf
        const float3 tminbound = (vol.bbox.Min() - c_w) / ray_w;
        const float3 tmaxbound = (vol.bbox.Max() - c_w) / ray_w;
        const float3 tmin = fminf(tminbound,tmaxbound);
        const float3 tmax = fmaxf(tminbound,tmaxbound);
        const float max_tmin = fmaxf(fmaxf(fmaxf(tmin.x, tmin.y), tmin.z), near);
        const float min_tmax = fminf(fminf(fminf(tmax.x, tmax.y), tmax.z), far);

        float depth = 0.0f;

        // If ray intersects bounding box
        if(max_tmin < min_tmax ) {
            // Go between max_tmin and min_tmax
            float lambda = max_tmin;
            float last_sdf = 0.0f/0.0f;
            float min_delta_lambda = vol.VoxelSizeUnits().x;
            float delta_lambda = 0;

            // March through space
            while(lambda < min_tmax) {
                const float3 pos_w = c_w + lambda * ray_w;
                const float sdf = vol.GetUnitsTrilinearClamped(pos_w);

                if( sdf <= 0 ) {
                    if( last_sdf > 0) {
                        // surface!
                        if(subpix) {
                            lambda = lambda + delta_lambda * sdf / (last_sdf - sdf);
                        }
                        depth = lambda;
                    }
                    break;
                }
                delta_lambda = sdf > 0 ? fmaxf(sdf, min_delta_lambda) : trunc_dist;
                lambda += delta_lambda;
                last_sdf = sdf;
            }
        }

        // Compute normal
        const float3 pos_w = c_w + depth * ray_w;
        const float3 _n_w = vol.GetUnitsBackwardDiffDxDyDz(pos_w);
        const float c = colorVol.GetUnitsTrilinearClamped(pos_w);
        const float len_n_w = length(_n_w);
        const float3 n_w = len_n_w > 0 ? _n_w / len_n_w : make_float3(0,0,1);
        const float3 n_c = mulSO3inv(T_wc,n_w);

        if(depth > 0 ) {
            imgdepth(u,v) = depth;
            img(u,v) = c;
            norm(u,v) = make_float4(n_c, 1);
        }else{
            imgdepth(u,v) = 0.0f/0.0f;
            img(u,v) = 0;
            norm(u,v) = make_float4(0,0,0,0);
        }
    }
}

void RaycastSdf(Image<float> depth, Image<float4> norm, Image<float> img, const BoundedVolume<SDF_t> vol, const BoundedVolume<float> colorVol, const Mat<float,3,4> T_wc, ImageIntrinsics K, float near, float far, float trunc_dist, bool subpix )
{
    dim3 blockDim, gridDim;
//    InitDimFromOutputImageOver(blockDim, gridDim, img, 16, 16);
    InitDimFromOutputImageOver(blockDim, gridDim, img);
    KernRaycastSdf<<<gridDim,blockDim>>>(depth, norm, img, vol, colorVol, T_wc, K, near, far, trunc_dist, subpix);
    GpuCheckErrors();
}

//////////////////////////////////////////////////////
// Raycast box
//////////////////////////////////////////////////////

__global__ void KernRaycastBox(Image<float> imgd, const Mat<float,3,4> T_wc, ImageIntrinsics K, const BoundingBox bbox )
{
    const int u = blockIdx.x*blockDim.x + threadIdx.x;
    const int v = blockIdx.y*blockDim.y + threadIdx.y;

    if( u < imgd.w && v < imgd.h ) {
        const float3 c_w = SE3Translation(T_wc);
        const float3 ray_c = K.Unproject(u,v);
        const float3 ray_w = mulSO3(T_wc, ray_c);

        // Raycast bounding box to find valid ray segment of sdf
        // http://www.cs.utah.edu/~awilliam/box/box.pdf
        const float3 tminbound = (bbox.Min() - c_w) / ray_w;
        const float3 tmaxbound = (bbox.Max() - c_w) / ray_w;
        const float3 tmin = fminf(tminbound,tmaxbound);
        const float3 tmax = fmaxf(tminbound,tmaxbound);
        const float max_tmin = fmaxf(fmaxf(tmin.x, tmin.y), tmin.z);
        const float min_tmax = fminf(fminf(tmax.x, tmax.y), tmax.z);

        float d;

        // If ray intersects bounding box
        if(max_tmin < min_tmax ) {
            d = max_tmin;
        }else{
            d = 0.0f/0.0f;
        }

        imgd(u,v) = d;
    }
}

void RaycastBox(Image<float> imgd, const Mat<float,3,4> T_wc, ImageIntrinsics K, const BoundingBox bbox )
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim, gridDim, imgd);
    KernRaycastBox<<<gridDim,blockDim>>>(imgd, T_wc, K, bbox);
    GpuCheckErrors();
}

//////////////////////////////////////////////////////
// Raycast sphere
//////////////////////////////////////////////////////

__global__ void KernRaycastSphere(Image<float> imgd, Image<float> img, ImageIntrinsics K, float3 center_c, float r)
{
    const int u = blockIdx.x*blockDim.x + threadIdx.x;
    const int v = blockIdx.y*blockDim.y + threadIdx.y;

    if( u < imgd.w && v < imgd.h ) {
        const float3 ray_c = K.Unproject(u,v);

        const float ldotc = dot(ray_c,center_c);
        const float lsq = dot(ray_c,ray_c);
        const float csq = dot(center_c,center_c);
        float depth = (ldotc - sqrt(ldotc*ldotc - lsq*(csq - r*r) )) / lsq;
        
        const float prev_depth = imgd(u,v);
        if(depth > 0 && (depth < prev_depth || !isfinite(prev_depth)) ) {
            imgd(u,v) = depth;
            if(img.ptr) {
                const float3 p_c = depth * ray_c;           
                const float3 n_c = p_c - center_c;
                img(u,v) = PhongShade(p_c, n_c / length(n_c));
            }
        }
    }
}

void RaycastSphere(Image<float> imgd, Image<float> img, const Mat<float,3,4> T_wc, ImageIntrinsics K, float3 center, float r)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim, gridDim, imgd);
    const float3 center_c = mulSE3inv(T_wc, center);    
    KernRaycastSphere<<<gridDim,blockDim>>>(imgd, img, K, center_c, r);
    GpuCheckErrors();
}

//////////////////////////////////////////////////////
// Raycast plane
//////////////////////////////////////////////////////

__global__ void KernRaycastPlane(Image<float> imgd, Image<float> img, ImageIntrinsics K, const float3 n_c)
{
    const int u = blockIdx.x*blockDim.x + threadIdx.x;
    const int v = blockIdx.y*blockDim.y + threadIdx.y;

    if( u < img.w && v < img.h ) {
        const float3 ray_c = K.Unproject(u,v);
        const float depth = -1 / dot(n_c, ray_c);

        const float prev_depth = imgd(u,v);
        if(depth > 0 && (depth < prev_depth || !isfinite(prev_depth)) ) {
            const float3 p_c = depth * ray_c;
            img(u,v) = PhongShade(p_c, n_c / length(n_c) );
            imgd(u,v) = depth;
        }
    }
}

void RaycastPlane(Image<float> imgd, Image<float> img, const Mat<float,3,4> T_wc, ImageIntrinsics K, const float3 n_w )
{
    const float3 n_c = Plane_b_from_a(T_wc, n_w);

    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim, gridDim, img);
    KernRaycastPlane<<<gridDim,blockDim>>>(imgd, img, K, n_c );
    GpuCheckErrors();
}


}
