#include "hip/hip_runtime.h"
#include "cu_convert.h"

#include "launch_utils.h"
#include "pixel_convert.h"

namespace roo
{

//////////////////////////////////////////////////////
// Image Conversion
//////////////////////////////////////////////////////

template<typename To, typename Ti>
__global__
void KernConvertImage(Image<To> dOut, const Image<Ti> dIn)
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    if(dOut.InBounds(x,y)) {
        dOut(x,y) = ConvertPixel<To,Ti>(dIn(x,y));
    }
}

template<typename To, typename Ti>
void ConvertImage(Image<To> dOut, const Image<Ti> dIn)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim, gridDim, dOut);
    KernConvertImage<<<gridDim,blockDim>>>(dOut,dIn);
}

// Explicit instantiation
template KANGAROO_EXPORT void ConvertImage<float,unsigned char>(Image<float>, const Image<unsigned char>);
template KANGAROO_EXPORT void ConvertImage<float,unsigned short>(Image<float>, const Image<unsigned short>);
template KANGAROO_EXPORT void ConvertImage<float,char>(Image<float>, const Image<char>);
template KANGAROO_EXPORT void ConvertImage<uchar4,uchar3>(Image<uchar4>, const Image<uchar3>);
template KANGAROO_EXPORT void ConvertImage<uchar3,uchar4>(Image<uchar3>, const Image<uchar4>);
template KANGAROO_EXPORT void ConvertImage<uchar3,unsigned char>(Image<uchar3>, const Image<unsigned char>);
template KANGAROO_EXPORT void ConvertImage<uchar4,unsigned char>(Image<uchar4>, const Image<unsigned char>);
template KANGAROO_EXPORT void ConvertImage<uchar4,float4>(Image<uchar4>, const Image<float4>);
template KANGAROO_EXPORT void ConvertImage<unsigned char, uchar3>(Image<unsigned char>, const Image<uchar3>);
template KANGAROO_EXPORT void ConvertImage<unsigned char, uchar4>(Image<unsigned char>, const Image<uchar4>);
template KANGAROO_EXPORT void ConvertImage<float4, float>(Image<float4>, const Image<float>);
template KANGAROO_EXPORT void ConvertImage<float4, uchar3>(Image<float4>, const Image<uchar3>);


} // namespace roo
