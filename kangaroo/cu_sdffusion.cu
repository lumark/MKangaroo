#include "hip/hip_runtime.h"
#include "cu_sdffusion.h"

#include "MatUtils.h"
#include "launch_utils.h"

namespace roo
{

//////////////////////////////////////////////////////
// Truncated SDF Fusion
// KinectFusion: Real-Time Dense Surface Mapping and Tracking, Newcombe et. al.
// http://www.doc.ic.ac.uk/~rnewcomb/
//////////////////////////////////////////////////////

__global__ void KernSdfFuse(BoundedVolume<SDF_t> vol, Image<float> depth,
                            Image<float4> normals, Mat<float,3,4> T_cw,
                            ImageIntrinsics K, float trunc_dist, float max_w, float mincostheta )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  const int z = blockIdx.z*blockDim.z + threadIdx.z;

  const float3 P_w = vol.VoxelPositionInUnits(x,y,z);
  const float3 P_c = T_cw * P_w;
  const float2 p_c = K.Project(P_c);

  if( depth.InBounds(p_c, 2) )
  {
    const float vd = P_c.z;
    //        const float md = depth.GetNearestNeighbour(p_c);
    //        const float3 mdn = make_float3(normals.GetNearestNeighbour(p_c));

    const float md = depth.GetBilinear<float>(p_c);
    const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

    const float costheta = dot(mdn, P_c) / -length(P_c);
    const float sd = costheta * (md - vd);
    const float w = costheta * 1.0f/vd;

    if(sd <= -trunc_dist) {
      // Further than truncation distance from surface
      // We do nothing.
    }else{
      //        }else if(sd < 5*trunc_dist) {
      if(isfinite(md) && isfinite(w) && costheta > mincostheta ) {
        SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
        sdf += vol(x,y,z);
        //                sdf.Clamp(-trunc_dist, trunc_dist);
        sdf.LimitWeight(max_w);
        vol(x,y,z) = sdf;
      }
    }
  }
}

void SdfFuse(BoundedVolume<SDF_t> vol, Image<float> depth, Image<float4> norm,
             Mat<float,3,4> T_cw, ImageIntrinsics K, float trunc_dist, float max_w, float mincostheta)
{
  dim3 blockDim(8,8,8);
  dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);
  KernSdfFuse<<<gridDim,blockDim>>>(vol, depth, norm, T_cw, K, trunc_dist, max_w, mincostheta);
  GpuCheckErrors();
}

//////////////////////////////////////////////////////
// Grey Truncated SDF Fusion
// Similar extension to KinectFusion as described by:
// Robust Tracking for Real-Time Dense RGB-D Mapping with Kintinous
// Whelan et. al.
//////////////////////////////////////////////////////

__global__ void KernSdfFuse(
    BoundedVolume<SDF_t> vol, BoundedVolume<float> colorVol,
    Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics K,
    Image<uchar3> img, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
    float trunc_dist, float max_w, float mincostheta
    )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  //    const int z = blockIdx.z*blockDim.z + threadIdx.z;
  for(int z=0; z < vol.d; ++z) {
    const float3 P_w = vol.VoxelPositionInUnits(x,y,z);
    const float3 P_c = T_cw * P_w;
    const float2 p_c = K.Project(P_c);
    const float3 P_i = T_iw * P_w;
    const float2 p_i = Kimg.Project(P_i);

    if( depth.InBounds(p_c, 2) && img.InBounds(p_i,2) )
    {
      const float vd = P_c.z;
      //            const float md = depth.GetNearestNeighbour(p_c);
      //            const float3 mdn = make_float3(normals.GetNearestNeighbour(p_c));
      //            const float c = ConvertPixel<float,uchar3>( img.GetNearestNeighbour(p_i) );

      const float md = depth.GetBilinear<float>(p_c);
      const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));
      const float c = ConvertPixel<float,float3>( img.GetBilinear<float3>(p_i) ) / 255.0;

      const float costheta = dot(mdn, P_c) / -length(P_c);
      const float sd = costheta * (md - vd);
      const float w = costheta * 1.0f/vd;

      if(sd <= -trunc_dist) {
        // Further than truncation distance from surface
        // We do nothing.
      }else{
        //        }else if(sd < 5*trunc_dist) {
        if(isfinite(md) && isfinite(w) && costheta > mincostheta ) {
          const SDF_t curvol = vol(x,y,z);
          SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
          sdf += curvol;
          sdf.LimitWeight(max_w);
          vol(x,y,z) = sdf;
          colorVol(x,y,z) = (w*c + colorVol(x,y,z) * curvol.w) / (w + curvol.w);
        }
      }
    }
  }
}

void SdfFuse(
    BoundedVolume<SDF_t> vol, BoundedVolume<float> colorVol,
    Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics K,
    Image<uchar3> img, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
    float trunc_dist, float max_w, float mincostheta
    ) {
  //    // 3d invoke
  //    dim3 blockDim(8,8,8);
  //    dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);
  //    KernSdfFuse<<<gridDim,blockDim>>>(vol, colorVol, depth, norm, T_cw, K, img, T_iw, Kimg, trunc_dist, max_w, mincostheta);
  //    GpuCheckErrors();

  dim3 blockDim(16,16);
  dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y);
  KernSdfFuse<<<gridDim,blockDim>>>(vol, colorVol, depth, norm, T_cw, K, img, T_iw, Kimg, trunc_dist, max_w, mincostheta);
  GpuCheckErrors();
}


//--the following add by luma-----------------------------------------------------------------------------------------------------------------------------
// do SDF fusion without consideing void (zero intensity) pixels
__global__ void KernSdfFuseDirectGrey(
    BoundedVolume<SDF_t> vol, BoundedVolume<float> colorVol,
    Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
    Image<float> grey, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
    float trunc_dist, float max_w, float mincostheta
    )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  //    const int z = blockIdx.z*blockDim.z + threadIdx.z;

  // For each voxel (x,y,z) we have in a bounded volume
  for(int z=0; z < vol.d; ++z)
  {
    // See if this voxel is possible to be in the image boundary
    // Get voxel position in certain radius in world coordinate
    const float3 P_w = vol.VoxelPositionInUnits(x,y,z);
    //        printf("loc:%f,%f,%f;",P_w.x,P_w.y,P_w.z);
    //    printf("x:%d,y:%d,z:%d,loc:%f,%f,%f;",x,y,z,P_w.x,P_w.y,P_w.z);

    // Get voxel position in camera coordinate
    const float3 P_c = T_cw * P_w;
    //    printf("P_c x:%f,y:%f,z:%f;",P_c.x,P_c.y,P_c.z);

    // Project a 3D voxel point to 2D depth an grey image coordinate
    const float2 p_c = Kdepth.Project(P_c);
    const float3 P_i = T_iw * P_w;
    const float2 p_i = Krgb.Project(P_i);

    // If the voxel is in image coordinate (inside of image boundary), then we
    // see if we should fuse this voxel
    if( depth.InBounds(p_c, 2) && grey.InBounds(p_i,2) )
    {
      // prepare to fuse a grey pixel into this voxel
      const float c =  grey.GetBilinear<float>(p_i);

      // discard pixel value equals 0
      if(c!=0)
      {
        // depth value at camera coorniate
        const float vd   = P_c.z;

        // depth value at image coordinate
        const float md   = depth.GetBilinear<float>(p_c);

        // normal value at image coordinate
        const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

        const float costheta = dot(mdn, P_c) / -length(P_c);
        const float sd = costheta * (md - vd);
        const float w = costheta * 1.0f/vd;

        if(sd <= -trunc_dist)
        {
          // Further than truncation distance from surface
          // We do nothing.
        }
        // update SDF
        else
        {
          //        }else if(sd < 5*trunc_dist) {

          /// here 0.5 is for kinect sensor
          if(/*sd < 5*trunc_dist && */isfinite(md) && md>0.5 && costheta > mincostheta )
          {
            //            printf("md %f,", md);
            const SDF_t curvol = vol(x,y,z);
            // return min of 'sd' and 'trunc_dist' as 'x', then rerurn max of 'x' and 'w'
            SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
            sdf += curvol;
            sdf.LimitWeight(max_w);
            vol(x,y,z) = sdf;
            colorVol(x,y,z) = (w*c + colorVol(x,y,z) * curvol.w) / (w + curvol.w);
          }
        }
      }
    }
  }
}

void SdfFuseDirectGrey(
    BoundedVolume<SDF_t> vol, BoundedVolume<float> colorVol,
    Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
    Image<float> grey, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
    float trunc_dist, float max_w, float mincostheta
    ) {
  dim3 blockDim(16,16);
  dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y);
  KernSdfFuseDirectGrey<<<gridDim,blockDim>>>(vol, colorVol, depth, norm, T_cw, Kdepth, grey, T_iw, Krgb, trunc_dist, max_w, mincostheta);
  GpuCheckErrors();
}



////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////// For Grid SDF Fusion /////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ BoundedVolumeGrid<SDF_t, roo::TargetDevice, roo::Manage>  g_Vol;
__device__ BoundedVolumeGrid<SDF_t, roo::TargetDevice, roo::Manage>  g_colorVol;

int GetAvailableGPUMemory()
{
  const unsigned bytes_per_mb = 1024*1000;
  size_t cu_mem_start, cu_mem_total;
  if(hipMemGetInfo( &cu_mem_start, &cu_mem_total ) != hipSuccess) {
    std::cerr << "Unable to get available memory" << std::endl;
    exit(-1);
  }

  int LeftMemory = cu_mem_start/bytes_per_mb;
  return LeftMemory;
}

// -----------------------------------------------------------------------------
//--the following add by luma---------------------------------------------------
// do SDF fusion without consideing void (zero intensity) pixels
__global__ void KernSdfFuseDirectGreyGrid(VolumeGrid<SDF_t, TargetDevice, Manage> testvol,
                                          Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
                                          Image<float> grey, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
                                          float trunc_dist, float max_w, float mincostheta
                                          )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  //    const int z = blockIdx.z*blockDim.z + threadIdx.z;

  // For each voxel (x,y,z) we have in a bounded volume
  for(int z=0; z < g_Vol.m_d; ++z)
  {
    // See if this voxel is possible to be in the image boundary
    // Get voxel position in certain radius in world coordinate (good)
    const float3 P_w = g_Vol.VoxelPositionInUnits(x,y,z);

    // Get voxel position in camera coordinate (good)
    const float3 P_c = T_cw * P_w;

    // Project a 3D voxel point to 2D depth an grey image coordinate
    const float2 p_c = Kdepth.Project(P_c);

    const float3 P_i = T_iw * P_w;
    const float2 p_i = Krgb.Project(P_i);


    // If the voxel is in image coordinate (inside of image boundary), then we
    // see if we should fuse this voxel
    if( depth.InBounds(p_c, 2) && grey.InBounds(p_i,2) )
    {

      //      printf("in bounds.");
      // prepare to fuse a grey pixel into this voxel
      const float c =  grey.GetBilinear<float>(p_i);

      // discard pixel value equals 0
      if(c!=0)
      {
        // depth value at camera coorniate
        const float vd   = P_c.z;

        // depth value at image coordinate
        const float md   = depth.GetBilinear<float>(p_c);

        // normal value at image coordinate
        const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

        const float costheta = dot(mdn, P_c) / -length(P_c);
        const float sd = costheta * (md - vd);
        const float w = costheta * 1.0f/vd;

        if(sd <= -trunc_dist)
        {
          // Further than truncation distance from surface
          // We do nothing.
        }
        // update SDF
        else
        {
          //        }else if(sd < 5*trunc_dist) {

          /// here 0.5 is for kinect sensor
          if(/*sd < 5*trunc_dist && */isfinite(md) && md>0.5 && costheta > mincostheta )
          {
            g_Vol.m_GridVolumes[0](0,0,0).val = g_Vol.m_GridVolumes[0](0,0,0).val +1;

            //            printf("val is %f",g_Vol.m_GridVolumes[0](0,0,0).val);

            printf("val is %f",testvol(0,0,0).val);


            //            printf("vol is %f, workspace is %f;",vol.m_dSingleVolumeGrid(0,0,0).val, GridSDFWorkSpace[0](0,0,0).val);

            //            printf("workspace is %f;", GridSDFWorkSpace[0](0,0,0).val);

            //            g_Vol.m_GridVolumes[0].Get(0,0,0).val = 1;

            //            printf("md %f,", md);
            //            printf("try to get vol value.");
            //            const SDF_t curvol = vol.GetElementSdf(x,y,z);

            //            const SDF_t curvol= vol.GetVal(x,y,z);

            // return min of 'sd' and 'trunc_dist' as 'x', then rerurn max of 'x' and 'w'
            //            SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
            //            sdf += curvol;
            //            sdf.LimitWeight(max_w);

            //            printf("val:%f",pTest->Get(0,0,0).val);
            //            printf("test num is:%d",testnum);

            /// set val
            //            vol.SetElement(sdf, x, y, z);
            //            vol.SetVal(sdf, x,y,z);
            //            printf("get:%f;",vol.GetVal(x,y,z).val);

            //            printf("get:%f",vol.m_hSingGridVolume->Get(0,0,0).val);

            //            vol.m_GridVolume[0][0][0]->Get(0,0,0) = sdf;
            //            g_colorVol.SetElement(x,y,z, (w*c + g_colorVol.GetElementFloat(x,y,z) * curvol.w) / (w + curvol.w));
            //            printf("fuse:%f,grey:%f,pc:%f,%f-pi:%f,%f;",sdf.val,c,p_c.x, p_c.y,p_i.x,p_i.y);
            //            printf("fuse:%f,grey:%f",sdf.val,colorVol.GetElementFloat(x,y,z));
            //            printf("fuse:%f",vol.m_GridVolume[0][0][0]->Get(0,0,0));
          }
        }
      }
    }
  }
}


void SdfFuseDirectGreyGrid(
    BoundedVolumeGrid<SDF_t, roo::TargetDevice, roo::Manage> vol,
    BoundedVolumeGrid<float, roo::TargetDevice, roo::Manage> colorVol,
    Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
    Image<float> grey, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
    float trunc_dist, float max_w, float mincostheta
    ) {

  //  int m_res = 4;

  //  BoundedVolume<SDF_t, TargetDevice, Manage>*   d_array;

  ////  int nSize = sizeof(vol.m_GridVolume);

  //  int nSize = sizeof(vol.m_TestVol) * 64;

  //  int nSize = sizeof(vol.m_hSingGridVolume);

  //  printf("each size is %d, %d",nSize);

  //  printf("get val %f",vol.m_hSingGridVolume->Get(0,0,0));

  //  // 1. Allocate device array.
  //  hipMalloc((void**) &(d_array), nSize);
  //  GpuCheckErrors();

  //  // 2. Copy array contents from host to device.
  //  hipMemcpy(d_array, vol.m_hSingGridVolume, nSize, hipMemcpyHostToDevice);
  //  GpuCheckErrors();

  // 3. Point to device pointer in host struct.
  //  for(int i=0;i!=4;i++)
  //  {
  //    for(int j=0;j!=4;j++)
  //    {
  //      for(int k=0;k!=4;k++)
  //      {
  //        vol.m_hGridVolume[i + j * m_res + k * m_res * m_res] = d_array[i + j * m_res + k * m_res * m_res];
  //      }
  //    }
  //  }

  //  vol.m_hSingGridVolume = d_array;

  //  d_array->CopyFrom

  //    BoundedVolume<SDF_t, TargetDevice, Manage>* d_array;

  //    int nSize = sizeof(vol.m_TestSDF);

  //    printf("each size is %d, %d",nSize);

  //    // 1. Allocate device array.
  //    hipMalloc((void**) &(d_array), nSize);
  //    GpuCheckErrors();

  //    // 2. Copy array contents from host to device.
  //    hipMemcpy(d_array, vol.m_GridVolume, nSize, hipMemcpyHostToDevice);
  //    GpuCheckErrors();


  hipMemcpyToSymbol(HIP_SYMBOL(g_Vol), &vol, sizeof(vol), size_t(0), hipMemcpyHostToDevice);
  GpuCheckErrors();

  // launch kernel
  dim3 blockDim(16,16);
  dim3 gridDim(vol.m_w / blockDim.x, vol.m_h / blockDim.y);
  KernSdfFuseDirectGreyGrid<<<gridDim,blockDim>>>(vol.m_GridVolumes[0], depth, norm, T_cw, Kdepth, grey, T_iw, Krgb, trunc_dist, max_w, mincostheta);
  GpuCheckErrors();

  // copy data back after launch the kernel
  for(int i=0;i!=64;i++)
  {
    vol.m_GridVolumes[i].MemcpyFromDevice(g_Vol.m_GridVolumes[i]);
  }
}



// -----------------------------------------------------------------------------
//--the following add by luma---------------------------------------------------
// do SDF fusion without consideing void (zero intensity) pixels
__global__ void KernSdfFuseColor(
    BoundedVolume<SDF_t> vol, BoundedVolume<uchar3> colorVol,
    Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics K,
    Image<float> img, Image<uchar3> Imgrgb, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
    float trunc_dist, float max_w, float mincostheta
    )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  //    const int z = blockIdx.z*blockDim.z + threadIdx.z;
  for(int z=0; z < vol.d; ++z)
  {
    const float3 P_w = vol.VoxelPositionInUnits(x,y,z);
    const float3 P_c = T_cw * P_w;
    const float2 p_c = K.Project(P_c);
    const float3 P_i = T_iw * P_w;
    const float2 p_i = Kimg.Project(P_i);

    if( depth.InBounds(p_c, 2) && img.InBounds(p_i,2) )
    {
      const float c =  img.GetBilinear<float>(p_i);

      // discard pixel value equals 0
      if(c!=0)
      {
        const float vd = P_c.z;
        const float md = depth.GetBilinear<float>(p_c);
        const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

        const float costheta = dot(mdn, P_c) / -length(P_c);
        const float sd = costheta * (md - vd);
        //                const float w = 1;
        const float w = costheta * 1.0f/vd;

        if(sd <= -trunc_dist)
        {
          // Further than truncation distance from surface
          // We do nothing.
        }
        // update SDF
        else
        {
          //        }else if(sd < 5*trunc_dist) {
          if(/*sd < 5*trunc_dist && */isfinite(md) && md>0.5 && costheta > mincostheta )
          {
            const SDF_t curvol = vol(x,y,z);
            // return min of 'sd' and 'trunc_dist' as 'x', then rerurn max of 'x' and 'w'
            SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
            sdf += curvol;
            sdf.LimitWeight(max_w);
            vol(x,y,z) = sdf;

            //            printf("(u,v)=(%d,%d),(r,g,b)=(%d,%d,%d),(x,y,z)=(%d,%d,%d)",int(p_i.x),int(p_i.y),
            //                   int(Imgrgb.Get(int(p_i.x),int(p_i.y)).x),
            //                   int(Imgrgb.Get(int(p_i.x),int(p_i.y)).y),
            //                   int(Imgrgb.Get(int(p_i.x),int(p_i.y)).z),
            //                   x,y,z);

            colorVol(x,y,z) = make_uchar3( Imgrgb.Get(int(p_i.x),int(p_i.y)).x,
                                           Imgrgb.Get(int(p_i.x),int(p_i.y)).y,
                                           Imgrgb.Get(int(p_i.x),int(p_i.y)).z) ;
          }
        }
      }
    }
  }
}

void SdfFuseColor(
    BoundedVolume<SDF_t> vol, BoundedVolume<uchar3> colorVol,
    Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics K,
    Image<float> img, Image<uchar3> Imgrgb, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
    float trunc_dist, float max_w, float mincostheta
    ) {
  dim3 blockDim(16,16);
  dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y);
  KernSdfFuseColor<<<gridDim,blockDim>>>(vol, colorVol, depth, norm,
                                         T_cw, K, img, Imgrgb, T_iw, Kimg,
                                         trunc_dist, max_w, mincostheta);
  GpuCheckErrors();
}

//-----------------------------------------------------------------------------------------------------------------------------------

//////////////////////////////////////////////////////
// find outline from sdf
/////////////////////////////////////////////////////
__global__ void KernSdfFuseFindOutline(
    BoundedVolume<SDF_t> vol,BoundedVolume<float> colorVol,
    Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics K,
    Image<uchar3> img, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
    float trunc_dist, float max_w, float mincostheta, Image<float4> dOutLine
    )
{

  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  //    const int z = blockIdx.z*blockDim.z + threadIdx.z;


  for(int z=0; z < vol.d; ++z)
  {
    const float3 P_w = vol.VoxelPositionInUnits(x,y,z);
    const float3 P_c = T_cw * P_w;
    const float2 p_c = K.Project(P_c);
    const float3 P_i = T_iw * P_w;
    const float2 p_i = Kimg.Project(P_i);

    // if voxel is inside bounds
    if( depth.InBounds(p_c, 2) && img.InBounds(p_i,2) )
    {
      const float vd = P_c.z;

      const float md = depth.GetBilinear<float>(p_c);
      const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));
      const float c = ConvertPixel<float,float3>( img.GetBilinear<float3>(p_i) ) / 255.0;

      const float costheta = dot(mdn, P_c) / -length(P_c);
      const float sd = costheta * (md - vd);
      const float w = costheta * 1.0f/vd;

      if(sd <= -trunc_dist)
      {
      }
      else
      {
        SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
        sdf += vol(x,y,z);

        sdf.LimitWeight(max_w);
        vol(x,y,z) = sdf;
      }
    }
    // out of boundary
    else
    {
      //            dOutLine(p_i.x,p_i.y) = make_float4(1, 0, 1, 1);
    }
  }

}




void SdfFuseFindOutline(
    BoundedVolume<SDF_t> vol, BoundedVolume<float> colorVol,Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics K,
    Image<uchar3> img, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
    float trunc_dist, float max_w, float mincostheta, Image<float4> dOutline
    ) {

  dim3 blockDim(16,16);
  dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y);
  KernSdfFuseFindOutline<<<gridDim,blockDim>>>(vol, colorVol, depth, norm, T_cw, K, img, T_iw, Kimg, trunc_dist, max_w, mincostheta, dOutline);
  GpuCheckErrors();
}


__global__ void KernFindBBBoundary(
    BoundedVolume<float> colorVol, int max_x, int max_y, int max_z, int min_x, int min_y, int min_z
    )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  //    const int z = blockIdx.z*blockDim.z + threadIdx.z;
  for(int z=0; z < colorVol.d; ++z)
  {
    const float curvol = colorVol(x,y,z);

    if(isfinite(curvol))
    {
      //            printf(",vol:%f", curvol);
      if(x>max_x)
      {
        max_x = x;
      }
      else if(x<min_x)
      {
        min_x = x;
      }

      if(y>max_y)
      {
        max_y = y;
      }
      else if(y<min_y)
      {
        min_y = y;
      }

      if(z>max_z)
      {
        max_z = z;
      }
      else if(z<min_z)
      {
        min_z = z;
      }
    }
  }
}

void FindBBBoundary(
    BoundedVolume<float> colorVol, int max_x, int max_y, int max_z, int min_x, int min_y, int min_z
    ) {
  dim3 blockDim(16,16);
  dim3 gridDim(colorVol.w / blockDim.x, colorVol.h / blockDim.y);
  KernFindBBBoundary<<<gridDim,blockDim>>>(colorVol, max_x, max_y, max_z, min_x, min_y, min_z);
  GpuCheckErrors();

}

// ---------------------------------------------------------------------------------------------------------------------------------


//////////////////////////////////////////////////////
// Reset SDF
//////////////////////////////////////////////////////

void SdfReset(BoundedVolume<SDF_t> vol, float trunc_dist)
{
  vol.Fill(SDF_t(0.0/0.0, 0));
}

void SdfReset(BoundedVolume<float> vol)
{
  vol.Fill(0.5);
}

void SdfReset(BoundedVolume<uchar3> vol)
{
  vol.Fill(make_uchar3( 0,0,0 ));
}

void SdfReset(VolumeGrid<SDF_t,roo::TargetDevice, roo::Manage> vol)
{
  vol.Fill(SDF_t(1, 1));
}

void SdfReset(VolumeGrid<float,roo::TargetDevice, roo::Manage> vol)
{
  vol.Fill(0.5);
}

//boxmin and boxmax define the box that is to be kept intact, rest will be cleared. This approach makes if conditions inside simpler.
//TODO: Name the function better.
__global__ void KernSdfResetPartial(BoundedVolume<SDF_t> vol, float3 boxmin, float3 boxmax)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  const int z = blockIdx.z*blockDim.z + threadIdx.z;

  const float3 P_w = vol.VoxelPositionInUnits(x,y,z);

  bool mincrit, maxcrit;//if mincrit and maxcrit are true, point is inside the box, i.e. valid.
  mincrit = P_w.x > boxmin.x && P_w.y < boxmax.y && P_w.z > boxmin.z;
  maxcrit = P_w.x < boxmax.x && P_w.y > boxmin.y && P_w.z < boxmax.z;

  if(!mincrit || !maxcrit)//i.e. the point is outside the box.
  {
    vol(x,y,z) = SDF_t(0.0/0.0,0.0);
  }
}

//TODO: Name the function better.
void SdfResetPartial(BoundedVolume<SDF_t> vol, float3 shift)
{
  //Initialization for GPU parallelization
  dim3 blockDim(8,8,8);
  dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);

  //compute the box to keep, it's conter intuitive to the name of function but more efficient.
  float3 bn = vol.bbox.boxmin, bx = vol.bbox.boxmax;//bn is box min and bx is box max.

  if(shift.x>0)
    bn.x += shift.x;
  else
    bx.x += shift.x;

  //y is -ve, but boxmax and boxmin for y are also inverse. i.e. the bottom most point is min.x,max.y,min.z
  if(shift.y>0)
    bn.y += shift.y;
  else
    bx.y += shift.y;

  if(shift.z>0)
    bn.z += shift.z;
  else
    bx.z += shift.z;

  KernSdfResetPartial<<<gridDim,blockDim>>>(vol, bn, bx);
  GpuCheckErrors();

}



//////////////////////////////////////////////////////
// Create SDF representation of sphere
//////////////////////////////////////////////////////

__global__ void KernSdfSphere(BoundedVolume<SDF_t> vol, float3 center, float r)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  const int z = blockIdx.z*blockDim.z + threadIdx.z;

  const float3 pos = vol.VoxelPositionInUnits(x,y,z);
  const float dist = length(pos - center);
  const float sdf = dist - r;

  vol(x,y,z) = SDF_t(sdf);
}

void SdfSphere(BoundedVolume<SDF_t> vol, float3 center, float r)
{
  dim3 blockDim(8,8,8);
  dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);

  KernSdfSphere<<<gridDim,blockDim>>>(vol, center, r);
  GpuCheckErrors();
}

//////////////////////////////////////////////////////
// Take SDF Difference to depthmap
//////////////////////////////////////////////////////

__global__ void KernSdfDistance(Image<float> dist, Image<float> depth, BoundedVolume<SDF_t> vol, const Mat<float,3,4> T_wc, ImageIntrinsics K, float trunc_distance)
{
  const int u = blockIdx.x*blockDim.x + threadIdx.x;
  const int v = blockIdx.y*blockDim.y + threadIdx.y;

  if( u < depth.w && v < depth.h ) {
    const float z = depth(u,v);
    const float3 p_c = z * K.Unproject(u,v);
    const float3 p_w = T_wc * p_c;

    const SDF_t sdf = vol.GetUnitsTrilinearClamped(p_w);
    dist(u,v) = sdf.val; //(sdf.val + trunc_distance) / (2* trunc_distance);
  }
}


void SdfDistance(Image<float> dist, Image<float> depth, BoundedVolume<SDF_t> vol, const Mat<float,3,4> T_wc, ImageIntrinsics K, float trunc_distance)
{
  dim3 blockDim, gridDim;
  InitDimFromOutputImageOver(blockDim, gridDim, depth);

  KernSdfDistance<<<gridDim,blockDim>>>(dist, depth, vol, T_wc, K, trunc_distance);
  GpuCheckErrors();
}

}
