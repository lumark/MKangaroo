#include "hip/hip_runtime.h"
#include "cu_sdffusion.h"

#include "MatUtils.h"
#include "launch_utils.h"

namespace roo
{

//////////////////////////////////////////////////////
// Truncated SDF Fusion
// KinectFusion: Real-Time Dense Surface Mapping and Tracking, Newcombe et. al.
// http://www.doc.ic.ac.uk/~rnewcomb/
//////////////////////////////////////////////////////

__global__ void KernSdfFuse(BoundedVolume<SDF_t> vol, Image<float> depth,
                            Image<float4> normals, Mat<float,3,4> T_cw,
                            ImageIntrinsics K, float trunc_dist, float max_w, float mincostheta )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  const int z = blockIdx.z*blockDim.z + threadIdx.z;

  const float3 P_w = vol.VoxelPositionInUnits(x,y,z);
  const float3 P_c = T_cw * P_w;
  const float2 p_c = K.Project(P_c);

  if( depth.InBounds(p_c, 2) )
  {
    const float vd = P_c.z;
    //        const float md = depth.GetNearestNeighbour(p_c);
    //        const float3 mdn = make_float3(normals.GetNearestNeighbour(p_c));

    const float md = depth.GetBilinear<float>(p_c);
    const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

    const float costheta = dot(mdn, P_c) / -length(P_c);
    const float sd = costheta * (md - vd);
    const float w = costheta * 1.0f/vd;

    if(sd <= -trunc_dist) {
      // Further than truncation distance from surface
      // We do nothing.
    }else{
      //        }else if(sd < 5*trunc_dist) {
      if(isfinite(md) && isfinite(w) && costheta > mincostheta ) {
        SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
        sdf += vol(x,y,z);
        //                sdf.Clamp(-trunc_dist, trunc_dist);
        sdf.LimitWeight(max_w);
        vol(x,y,z) = sdf;
      }
    }
  }
}

void SdfFuse(BoundedVolume<SDF_t> vol, Image<float> depth, Image<float4> norm,
             Mat<float,3,4> T_cw, ImageIntrinsics K, float trunc_dist, float max_w, float mincostheta)
{
  dim3 blockDim(8,8,8);
  dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);
  KernSdfFuse<<<gridDim,blockDim>>>(vol, depth, norm, T_cw, K, trunc_dist, max_w, mincostheta);
  GpuCheckErrors();
}

//////////////////////////////////////////////////////
// gray Truncated SDF Fusion
// Similar extension to KinectFusion as described by:
// Robust Tracking for Real-Time Dense RGB-D Mapping with Kintinous
// Whelan et. al.
//////////////////////////////////////////////////////

__global__ void KernSdfFuse(
    BoundedVolume<SDF_t> vol, BoundedVolume<float> colorVol,
    Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics K,
    Image<uchar3> img, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
    float trunc_dist, float max_w, float mincostheta
    )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  //    const int z = blockIdx.z*blockDim.z + threadIdx.z;
  for(int z=0; z < vol.d; ++z) {
    const float3 P_w = vol.VoxelPositionInUnits(x,y,z);
    const float3 P_c = T_cw * P_w;
    const float2 p_c = K.Project(P_c);
    const float3 P_i = T_iw * P_w;
    const float2 p_i = Kimg.Project(P_i);

    if( depth.InBounds(p_c, 2) && img.InBounds(p_i,2) )
    {
      const float vd = P_c.z;
      const float md = depth.GetBilinear<float>(p_c);
      const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));
      const float c = ConvertPixel<float,float3>( img.GetBilinear<float3>(p_i) ) / 255.0;

      const float costheta = dot(mdn, P_c) / -length(P_c);
      const float sd = costheta * (md - vd);
      const float w = costheta * 1.0f/vd;

      if(sd <= -trunc_dist) {
        // Further than truncation distance from surface
        // We do nothing.
      }else{
        //        }else if(sd < 5*trunc_dist) {
        if(isfinite(md) && isfinite(w) && costheta > mincostheta ) {
          const SDF_t curvol = vol(x,y,z);
          SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
          sdf += curvol;
          sdf.LimitWeight(max_w);
          vol(x,y,z) = sdf;
          colorVol(x,y,z) = (w*c + colorVol(x,y,z) * curvol.w) / (w + curvol.w);
        }
      }
    }
  }
}

void SdfFuse(
    BoundedVolume<SDF_t> vol, BoundedVolume<float> colorVol,
    Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics K,
    Image<uchar3> img, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
    float trunc_dist, float max_w, float mincostheta
    )
{
  dim3 blockDim(16,16);
  dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y);
  KernSdfFuse<<<gridDim,blockDim>>>(vol, colorVol, depth, norm, T_cw, K, img, T_iw, Kimg, trunc_dist, max_w, mincostheta);
  GpuCheckErrors();
}


//--the following add by luma-----------------------------------------------------------------------------------------------------------------------------
// do SDF fusion without consideing void (zero intensity) pixels
__global__ void KernSdfFuseDirectGray(
    BoundedVolume<SDF_t> vol, BoundedVolume<float> colorVol,
    Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
    Image<float> gray, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
    float trunc_dist, float max_w, float mincostheta
    )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  //    const int z = blockIdx.z*blockDim.z + threadIdx.z;

  // For each voxel (x,y,z) we have in a bounded volume
  for(int z=0; z < vol.d; ++z)
  {
    // See if this voxel is possible to be in the image boundary
    // Get voxel position in certain radius in world coordinate
    const float3 P_w = vol.VoxelPositionInUnits(x,y,z);

    // Get voxel position in camera coordinate
    const float3 P_c = T_cw * P_w;

    // Project a 3D voxel point to 2D depth an gray image coordinate
    const float2 p_c = Kdepth.Project(P_c);
    const float3 P_i = T_iw * P_w;
    const float2 p_i = Krgb.Project(P_i);

    // If the voxel is in image coordinate (inside of image boundary), then we
    // see if we should fuse this voxel
    if( depth.InBounds(p_c, 2) && gray.InBounds(p_i,2) )
    {
      // prepare to fuse a gray pixel into this voxel
      const float c =  gray.GetBilinear<float>(p_i);

      // discard pixel value equals 0
      if(c!=0)
      {
        // depth value at camera coorniate
        const float vd   = P_c.z;

        // depth value at image coordinate
        const float md   = depth.GetBilinear<float>(p_c);

        // normal value at image coordinate
        const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

        const float costheta = dot(mdn, P_c) / -length(P_c);
        const float sd = costheta * (md - vd);
        const float w = costheta * 1.0f/vd;

        if(sd <= -trunc_dist)
        {
          // Further than truncation distance from surface
          // We do nothing.
        }
        // update SDF
        else
        {
          //        }else if(sd < 5*trunc_dist) {

          /// here 0.5 is for kinect sensor
          if(/*sd < 5*trunc_dist && */isfinite(md) && md>0.5 && costheta > mincostheta )
          {
            //            printf("md %f,", md);
            const SDF_t curvol = vol(x,y,z);
            // return min of 'sd' and 'trunc_dist' as 'x', then rerurn max of 'x' and 'w'
            SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
            sdf += curvol;
            sdf.LimitWeight(max_w);
            vol(x,y,z) = sdf;
            colorVol(x,y,z) = (w*c + colorVol(x,y,z) * curvol.w) / (w + curvol.w);
          }
        }
      }
    }
  }
}

void SdfFuseDirectGray(
    BoundedVolume<SDF_t> vol, BoundedVolume<float> colorVol,
    Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
    Image<float> gray, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
    float trunc_dist, float max_w, float mincostheta
    ) {
  dim3 blockDim(16,16);
  dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y);
  KernSdfFuseDirectGray<<<gridDim,blockDim>>>(vol, colorVol, depth, norm, T_cw, Kdepth, gray, T_iw, Krgb, trunc_dist, max_w, mincostheta);
  GpuCheckErrors();
}


// -----------------------------------------------------------------------------
//--the following add by luma---------------------------------------------------
// do SDF fusion without consideing void (zero intensity) pixels
__global__ void KernSdfFuseColor(
    BoundedVolume<SDF_t> vol, BoundedVolume<uchar3> colorVol,
    Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics K,
    Image<float> img, Image<uchar3> Imgrgb, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
    float trunc_dist, float max_w, float mincostheta
    )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  //    const int z = blockIdx.z*blockDim.z + threadIdx.z;
  for(int z=0; z < vol.d; ++z)
  {
    const float3 P_w = vol.VoxelPositionInUnits(x,y,z);
    const float3 P_c = T_cw * P_w;
    const float2 p_c = K.Project(P_c);
    const float3 P_i = T_iw * P_w;
    const float2 p_i = Kimg.Project(P_i);

    if( depth.InBounds(p_c, 2) && img.InBounds(p_i,2) )
    {
      const float c =  img.GetBilinear<float>(p_i);

      // discard pixel value equals 0
      if(c!=0)
      {
        const float vd = P_c.z;
        const float md = depth.GetBilinear<float>(p_c);
        const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

        const float costheta = dot(mdn, P_c) / -length(P_c);
        const float sd = costheta * (md - vd);
        //                const float w = 1;
        const float w = costheta * 1.0f/vd;

        if(sd <= -trunc_dist)
        {
          // Further than truncation distance from surface
          // We do nothing.
        }
        // update SDF
        else
        {
          //        }else if(sd < 5*trunc_dist) {
          if(/*sd < 5*trunc_dist && */isfinite(md) && md>0.5 && costheta > mincostheta )
          {
            const SDF_t curvol = vol(x,y,z);
            // return min of 'sd' and 'trunc_dist' as 'x', then rerurn max of 'x' and 'w'
            SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
            sdf += curvol;
            sdf.LimitWeight(max_w);
            vol(x,y,z) = sdf;

            //            printf("(u,v)=(%d,%d),(r,g,b)=(%d,%d,%d),(x,y,z)=(%d,%d,%d)",int(p_i.x),int(p_i.y),
            //                   int(Imgrgb.Get(int(p_i.x),int(p_i.y)).x),
            //                   int(Imgrgb.Get(int(p_i.x),int(p_i.y)).y),
            //                   int(Imgrgb.Get(int(p_i.x),int(p_i.y)).z),
            //                   x,y,z);

            colorVol(x,y,z) = make_uchar3( Imgrgb.Get(int(p_i.x),int(p_i.y)).x,
                                           Imgrgb.Get(int(p_i.x),int(p_i.y)).y,
                                           Imgrgb.Get(int(p_i.x),int(p_i.y)).z) ;
          }
        }
      }
    }
  }
}

void SdfFuseColor(
    BoundedVolume<SDF_t> vol, BoundedVolume<uchar3> colorVol,
    Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics K,
    Image<float> img, Image<uchar3> Imgrgb, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
    float trunc_dist, float max_w, float mincostheta
    ) {
  dim3 blockDim(16,16);
  dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y);
  KernSdfFuseColor<<<gridDim,blockDim>>>(vol, colorVol, depth, norm,
                                         T_cw, K, img, Imgrgb, T_iw, Kimg,
                                         trunc_dist, max_w, mincostheta);
  GpuCheckErrors();
}


//////////////////////////////////////////////////////
// Reset SDF
//////////////////////////////////////////////////////



void SdfReset(BoundedVolume<SDF_t> vol, float trunc_dist)
{
  vol.Fill(SDF_t(0.0/0.0, 0));
}

void SdfReset(BoundedVolume<float> vol)
{
  vol.Fill(0.5);
}

void SdfReset(BoundedVolume<uchar3> vol)
{
  vol.Fill(make_uchar3( 0,0,0 ));
}




//////////////////////////////////////////////////////
// Create SDF representation of sphere
//////////////////////////////////////////////////////

__global__ void KernSdfSphere(BoundedVolume<SDF_t> vol, float3 center, float r)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  const int z = blockIdx.z*blockDim.z + threadIdx.z;

  const float3 pos = vol.VoxelPositionInUnits(x,y,z);
  const float dist = length(pos - center);
  const float sdf = dist - r;

  vol(x,y,z) = SDF_t(sdf);
}

void SdfSphere(BoundedVolume<SDF_t> vol, float3 center, float r)
{
  dim3 blockDim(8,8,8);
  dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);

  KernSdfSphere<<<gridDim,blockDim>>>(vol, center, r);
  GpuCheckErrors();
}

//////////////////////////////////////////////////////
// Take SDF Difference to depthmap
//////////////////////////////////////////////////////

__global__ void KernSdfDistance(Image<float> dist, Image<float> depth, BoundedVolume<SDF_t> vol, const Mat<float,3,4> T_wc, ImageIntrinsics K, float trunc_distance)
{
  const int u = blockIdx.x*blockDim.x + threadIdx.x;
  const int v = blockIdx.y*blockDim.y + threadIdx.y;

  if( u < depth.w && v < depth.h ) {
    const float z = depth(u,v);
    const float3 p_c = z * K.Unproject(u,v);
    const float3 p_w = T_wc * p_c;

    const SDF_t sdf = vol.GetUnitsTrilinearClamped(p_w);
    dist(u,v) = sdf.val; //(sdf.val + trunc_distance) / (2* trunc_distance);
  }
}


void SdfDistance(Image<float> dist, Image<float> depth, BoundedVolume<SDF_t> vol, const Mat<float,3,4> T_wc, ImageIntrinsics K, float trunc_distance)
{
  dim3 blockDim, gridDim;
  InitDimFromOutputImageOver(blockDim, gridDim, depth);

  KernSdfDistance<<<gridDim,blockDim>>>(dist, depth, vol, T_wc, K, trunc_distance);
  GpuCheckErrors();
}

}
