#include "hip/hip_runtime.h"
#include "cu_sdffusion.h"

#include "MatUtils.h"
#include "launch_utils.h"

namespace roo
{

//////////////////////////////////////////////////////
// Truncated SDF Fusion
// KinectFusion: Real-Time Dense Surface Mapping and Tracking, Newcombe et. al.
// http://www.doc.ic.ac.uk/~rnewcomb/
//////////////////////////////////////////////////////

__global__ void KernSdfFuse(BoundedVolume<SDF_t> vol, Image<float> depth,
                            Image<float4> normals, Mat<float,3,4> T_cw,
                            ImageIntrinsics K, float trunc_dist, float max_w, float mincostheta )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  const int z = blockIdx.z*blockDim.z + threadIdx.z;

  const float3 P_w = vol.VoxelPositionInUnits(x,y,z);
  const float3 P_c = T_cw * P_w;
  const float2 p_c = K.Project(P_c);

  if( depth.InBounds(p_c, 2) )
  {
    const float vd = P_c.z;
    //        const float md = depth.GetNearestNeighbour(p_c);
    //        const float3 mdn = make_float3(normals.GetNearestNeighbour(p_c));

    const float md = depth.GetBilinear<float>(p_c);
    const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

    const float costheta = dot(mdn, P_c) / -length(P_c);
    const float sd = costheta * (md - vd);
    const float w = costheta * 1.0f/vd;

    if(sd <= -trunc_dist) {
      // Further than truncation distance from surface
      // We do nothing.
    }else{
      //        }else if(sd < 5*trunc_dist) {
      if(isfinite(md) && isfinite(w) && costheta > mincostheta ) {
        SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
        sdf += vol(x,y,z);
        //                sdf.Clamp(-trunc_dist, trunc_dist);
        sdf.LimitWeight(max_w);
        vol(x,y,z) = sdf;
      }
    }
  }
}

void SdfFuse(BoundedVolume<SDF_t> vol, Image<float> depth, Image<float4> norm,
             Mat<float,3,4> T_cw, ImageIntrinsics K, float trunc_dist, float max_w, float mincostheta)
{
  dim3 blockDim(8,8,8);
  dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);
  KernSdfFuse<<<gridDim,blockDim>>>(vol, depth, norm, T_cw, K, trunc_dist, max_w, mincostheta);
  GpuCheckErrors();
}

//////////////////////////////////////////////////////
// gray Truncated SDF Fusion
// Similar extension to KinectFusion as described by:
// Robust Tracking for Real-Time Dense RGB-D Mapping with Kintinous
// Whelan et. al.
//////////////////////////////////////////////////////

__global__ void KernSdfFuse(
    BoundedVolume<SDF_t> vol, BoundedVolume<float> colorVol,
    Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics K,
    Image<uchar3> img, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
    float trunc_dist, float max_w, float mincostheta
    )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  //    const int z = blockIdx.z*blockDim.z + threadIdx.z;
  for(int z=0; z < vol.d; ++z) {
    const float3 P_w = vol.VoxelPositionInUnits(x,y,z);
    const float3 P_c = T_cw * P_w;
    const float2 p_c = K.Project(P_c);
    const float3 P_i = T_iw * P_w;
    const float2 p_i = Kimg.Project(P_i);

    if( depth.InBounds(p_c, 2) && img.InBounds(p_i,2) )
    {
      const float vd = P_c.z;
      const float md = depth.GetBilinear<float>(p_c);
      const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));
      const float c = ConvertPixel<float,float3>( img.GetBilinear<float3>(p_i) ) / 255.0;

      const float costheta = dot(mdn, P_c) / -length(P_c);
      const float sd = costheta * (md - vd);
      const float w = costheta * 1.0f/vd;

      if(sd <= -trunc_dist) {
        // Further than truncation distance from surface
        // We do nothing.
      }else{
        //        }else if(sd < 5*trunc_dist) {
        if(isfinite(md) && isfinite(w) && costheta > mincostheta ) {
          const SDF_t curvol = vol(x,y,z);
          SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
          sdf += curvol;
          sdf.LimitWeight(max_w);
          vol(x,y,z) = sdf;
          colorVol(x,y,z) = (w*c + colorVol(x,y,z) * curvol.w) / (w + curvol.w);
        }
      }
    }
  }
}

void SdfFuse(
    BoundedVolume<SDF_t> vol, BoundedVolume<float> colorVol,
    Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics K,
    Image<uchar3> img, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
    float trunc_dist, float max_w, float mincostheta
    )
{
  dim3 blockDim(16,16);
  dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y);
  KernSdfFuse<<<gridDim,blockDim>>>(vol, colorVol, depth, norm, T_cw, K, img, T_iw, Kimg, trunc_dist, max_w, mincostheta);
  GpuCheckErrors();
}


//--the following add by luma-----------------------------------------------------------------------------------------------------------------------------
// do SDF fusion without consideing void (zero intensity) pixels
__global__ void KernSdfFuseDirectGray(
    BoundedVolume<SDF_t> vol, BoundedVolume<float> colorVol,
    Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
    Image<float> gray, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
    float trunc_dist, float max_w, float mincostheta
    )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  //    const int z = blockIdx.z*blockDim.z + threadIdx.z;

  // For each voxel (x,y,z) we have in a bounded volume
  for(int z=0; z < vol.d; ++z)
  {
    // See if this voxel is possible to be in the image boundary
    // Get voxel position in certain radius in world coordinate
    const float3 P_w = vol.VoxelPositionInUnits(x,y,z);

    // Get voxel position in camera coordinate
    const float3 P_c = T_cw * P_w;

    // Project a 3D voxel point to 2D depth an gray image coordinate
    const float2 p_c = Kdepth.Project(P_c);
    const float3 P_i = T_iw * P_w;
    const float2 p_i = Krgb.Project(P_i);

    // If the voxel is in image coordinate (inside of image boundary), then we
    // see if we should fuse this voxel
    if( depth.InBounds(p_c, 2) && gray.InBounds(p_i,2) )
    {
      // prepare to fuse a gray pixel into this voxel
      const float c =  gray.GetBilinear<float>(p_i);

      // discard pixel value equals 0
      if(c!=0)
      {
        // depth value at camera coorniate
        const float vd   = P_c.z;

        // depth value at image coordinate
        const float md   = depth.GetBilinear<float>(p_c);

        // normal value at image coordinate
        const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

        const float costheta = dot(mdn, P_c) / -length(P_c);
        const float sd = costheta * (md - vd);
        const float w = costheta * 1.0f/vd;

        if(sd <= -trunc_dist)
        {
          // Further than truncation distance from surface
          // We do nothing.
        }
        // update SDF
        else
        {
          //        }else if(sd < 5*trunc_dist) {

          /// here 0.5 is for kinect sensor
          if(/*sd < 5*trunc_dist && */isfinite(md) && md>0.5 && costheta > mincostheta )
          {
            //            printf("md %f,", md);
            const SDF_t curvol = vol(x,y,z);
            // return min of 'sd' and 'trunc_dist' as 'x', then rerurn max of 'x' and 'w'
            SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
            sdf += curvol;
            sdf.LimitWeight(max_w);
            vol(x,y,z) = sdf;
            colorVol(x,y,z) = (w*c + colorVol(x,y,z) * curvol.w) / (w + curvol.w);
          }
        }
      }
    }
  }
}

void SdfFuseDirectGray(
    BoundedVolume<SDF_t> vol, BoundedVolume<float> colorVol,
    Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
    Image<float> gray, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
    float trunc_dist, float max_w, float mincostheta
    ) {
  dim3 blockDim(16,16);
  dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y);
  KernSdfFuseDirectGray<<<gridDim,blockDim>>>(vol, colorVol, depth, norm, T_cw, Kdepth, gray, T_iw, Krgb, trunc_dist, max_w, mincostheta);
  GpuCheckErrors();
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////// For Grid SDF Fusion /////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ BoundedVolumeGrid<SDF_t, roo::TargetDevice, roo::Manage>  g_vol;
__device__ BoundedVolumeGrid<float, roo::TargetDevice, roo::Manage>  g_grayVol;

// have a large size of array to save index of grid sdf that need to init
__device__ int                                                       g_NextInitSDFs[102400];

// -----------------------------------------------------------------------------
//--the following add by luma---------------------------------------------------
// do SDF fusion without consideing void (zero intensity) pixels
__global__ void KernSdfInitGrayGrid(
    Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
    Image<float> gray, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
    float trunc_dist, float max_w, float mincostheta
    )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  //    const int z = blockIdx.z*blockDim.z + threadIdx.z;

  // For each voxel (x,y,z) we have in a bounded volume
  for(int z=0; z < g_vol.m_d; ++z)
  {
    // See if this voxel is possible to be in the image boundary
    // Get voxel position in certain radius in world coordinate (good)
    const float3 P_w = g_vol.VoxelPositionInUnits(x,y,z);

    // Get voxel position in camera coordinate (good)
    const float3 P_c = T_cw * P_w;

    // Project a 3D voxel point to 2D depth an gray image coordinate
    const float2 p_c = Kdepth.Project(P_c);

    const float3 P_i = T_iw * P_w;
    const float2 p_i = Krgb.Project(P_i);

    // If the voxel is in image coordinate (inside of image boundary), then we
    // see if we should fuse this voxel
    if( depth.InBounds(p_c, 2) && gray.InBounds(p_i,2) )
    {
      // prepare to fuse a gray pixel into this voxel
      const float c =  gray.GetBilinear<float>(p_i);

      // discard pixel value equals 0
      if(c>0)
      {
        // depth value at camera coorniate
        const float vd   = P_c.z;

        // depth value at image coordinate
        const float md   = depth.GetBilinear<float>(p_c);

        if(md>0.5)
        {
          // normal value at image coordinate
          const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

          const float costheta = dot(mdn, P_c) / -length(P_c);
          const float sd = costheta * (md - vd);

          if(sd <= -trunc_dist)
          {
            // Further than truncation distance from surface
            // We do nothing.
          }
          // update SDF
          else
          {
            //        }else if(sd < 5*trunc_dist) {
            /// here 0.5 is for kinect sensor
            if(/*sd < 5*trunc_dist && */isfinite(md)  && costheta > mincostheta )
            {
              int nIndex = g_vol.GetIndex(int(floorf(x/g_vol.m_nVolumeGridRes)),
                                          int(floorf(y/g_vol.m_nVolumeGridRes)),
                                          int(floorf(z/g_vol.m_nVolumeGridRes)) );
              g_NextInitSDFs[nIndex] = 1;
            }
          }
        }
      }
    }
  }
}


void SDFInitGrayGrid( int* pNextInitSDFs,
                      BoundedVolumeGrid<SDF_t, roo::TargetDevice, roo::Manage> vol,
                      BoundedVolumeGrid<float, roo::TargetDevice, roo::Manage> grayVol,
                      Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
                      Image<float> gray, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
                      float trunc_dist, float max_w, float mincostheta
                      )
{
  if(vol.m_nWholeGridRes*vol.m_nWholeGridRes*vol.m_nWholeGridRes>102400)
  {
    printf("[SDFInitgrayGrid.cu] Fatal Error! Array size overflow!\n");
    exit(-1);
  }

  // load grid sdf to golbal memory. We do this because there is a size limit of
  // the parameters that we can send the the kernel function.
  hipMemcpyToSymbol(HIP_SYMBOL(g_vol), &vol, sizeof(vol), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(g_grayVol), &grayVol, sizeof(grayVol), size_t(0), hipMemcpyHostToDevice);
  GpuCheckErrors();

  // launch kernel for SDF fusion
  dim3 blockDim(32,32);
  dim3 gridDim(vol.m_w / blockDim.x, vol.m_h / blockDim.y);
  KernSdfInitGrayGrid<<<gridDim,blockDim>>>(depth, norm, T_cw, Kdepth, gray, T_iw, Krgb, trunc_dist, max_w, mincostheta);
  GpuCheckErrors();

  //  printf("[SDFInitgrayGrid.cu] Finished kernel.\n");

  int nNextInitSDFs[102400];
  hipMemcpyFromSymbol(nNextInitSDFs, HIP_SYMBOL(g_NextInitSDFs), sizeof(g_NextInitSDFs), 0, hipMemcpyDeviceToHost);
  GpuCheckErrors();

  //  printf("[SDFInitgrayGrid.cu] Finished copy.\n");

  // copy array back
  for(int i=0;i!=vol.m_nWholeGridRes*vol.m_nWholeGridRes*vol.m_nWholeGridRes;i++)
  {
    pNextInitSDFs[i] = nNextInitSDFs[i];
    nNextInitSDFs[i] = 0;
  }

  // reset index
  hipMemcpyToSymbol(HIP_SYMBOL(g_NextInitSDFs),nNextInitSDFs,sizeof(nNextInitSDFs),0,hipMemcpyHostToDevice);
  GpuCheckErrors();

  // cuda free memory
  g_vol.FreeMemory();
  g_grayVol.FreeMemory();
  GpuCheckErrors();

  //  free(nNextInitSDFs);

  //  printf("[SDFInitgrayGrid.cu] Finished.\n");
}

// -----------------------------------------------------------------------------
//--the following add by luma---------------------------------------------------
// do SDF fusion without consideing void (zero intensity) pixels
// the following must be used with SDFInitgrayGrid
__global__ void KernSdfFuseDirectGrayGrid(
    Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
    Image<float> gray, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
    float trunc_dist, float max_w, float mincostheta
    )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  //    const int z = blockIdx.z*blockDim.z + threadIdx.z;

  // For each voxel (x,y,z) we have in a bounded volume
  for(int z=0; z < g_vol.m_d; ++z)
  {
    // See if this voxel is possible to be in the image boundary
    // Get voxel position in certain radius in world coordinate (good)
    const float3 P_w = g_vol.VoxelPositionInUnits(x,y,z);

    // Get voxel position in camera coordinate (good)
    const float3 P_c = T_cw * P_w;

    // Project a 3D voxel point to 2D depth an gray image coordinate
    const float2 p_c = Kdepth.Project(P_c);

    const float3 P_i = T_iw * P_w;
    const float2 p_i = Krgb.Project(P_i);

    // If the voxel is in image coordinate (inside of image boundary), then we
    // see if we should fuse this voxel
    if( depth.InBounds(p_c, 2) && gray.InBounds(p_i,2) )
    {
      // prepare to fuse a gray pixel into this voxel
      const float c =  gray.GetBilinear<float>(p_i);

      // discard pixel value equals 0
      if(c!=0)
      {
        // depth value at camera coorniate
        const float vd   = P_c.z;

        // depth value at image coordinate
        const float md   = depth.GetBilinear<float>(p_c);

        // normal value at image coordinate
        const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

        const float costheta = dot(mdn, P_c) / -length(P_c);
        const float sd = costheta * (md - vd);
        const float w = costheta * 1.0f/vd;

        if(sd <= -trunc_dist)
        {
          // Further than truncation distance from surface
          // We do nothing.
        }
        // update SDF
        else
        {
          //        }else if(sd < 5*trunc_dist) {

          /// here 0.5 is for kinect sensor
          if(/*sd < 5*trunc_dist && */isfinite(md) && md>0.5 && costheta > mincostheta )
          {
            const SDF_t curvol = g_vol(x,y,z);

            // return min of 'sd' and 'trunc_dist' as 'x', then rerurn max of 'x' and 'w'
            SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
            sdf += curvol;
            sdf.LimitWeight(max_w);

            /// set val
            g_vol(x, y, z) = sdf;
            g_grayVol(x,y,z) = (w*c + g_grayVol(x,y,z) * curvol.w) / (w + curvol.w);
          }
        }
      }
    }
  }
}


void SdfFuseDirectGrayGrid(
    BoundedVolumeGrid<SDF_t, roo::TargetDevice, roo::Manage> vol,
    BoundedVolumeGrid<float, roo::TargetDevice, roo::Manage> colorVol,
    Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
    Image<float> gray, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
    float trunc_dist, float max_w, float mincostheta
    )
{
  /// load grid sdf to golbal memory. We do this because there is a size limit of
  // the parameters that we can send the the kernel function.
  hipMemcpyToSymbol(HIP_SYMBOL(g_vol), &vol, sizeof(vol), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(g_grayVol), &colorVol, sizeof(colorVol), size_t(0), hipMemcpyHostToDevice);
  GpuCheckErrors();

  // launch kernel for SDF fusion
  dim3 blockDim(32,32);
  dim3 gridDim(vol.m_w / blockDim.x, vol.m_h / blockDim.y);
  KernSdfFuseDirectGrayGrid<<<gridDim,blockDim>>>(depth, norm, T_cw, Kdepth, gray, T_iw, Krgb, trunc_dist, max_w, mincostheta);
  GpuCheckErrors();

  // copy data back after launch the kernel
  vol.CopyFrom(g_vol);
  colorVol.CopyFrom(g_grayVol);
  GpuCheckErrors();

  // cuda free memory
  g_vol.FreeMemory();
  g_grayVol.FreeMemory();
  GpuCheckErrors();
}



// -----------------------------------------------------------------------------
//--the following add by luma---------------------------------------------------
// do SDF fusion without consideing void (zero intensity) pixels
// the following must be used with SDFInitgrayGrid
__global__ void KernSdfFuseDirectGrayGridSafe(
    Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
    Image<float> gray, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
    float trunc_dist, float max_w, float mincostheta
    )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  //    const int z = blockIdx.z*blockDim.z + threadIdx.z;

  // For each voxel (x,y,z) we have in a bounded volume
  for(int z=0; z < g_vol.m_d; ++z)
  {
    // See if this voxel is possible to be in the image boundary
    // Get voxel position in certain radius in world coordinate (good)
    const float3 P_w = g_vol.VoxelPositionInUnits(x,y,z);

    // Get voxel position in camera coordinate (good)
    const float3 P_c = T_cw * P_w;

    // Project a 3D voxel point to 2D depth an gray image coordinate
    const float2 p_c = Kdepth.Project(P_c);

    const float3 P_i = T_iw * P_w;
    const float2 p_i = Krgb.Project(P_i);

    // If the voxel is in image coordinate (inside of image boundary), then we
    // see if we should fuse this voxel
    if( depth.InBounds(p_c, 2) && gray.InBounds(p_i,2) )
    {
      // prepare to fuse a gray pixel into this voxel
      const float c =  gray.GetBilinear<float>(p_i);

      // discard pixel value equals 0
      if(c!=0)
      {
        // depth value at camera coorniate
        const float vd   = P_c.z;

        // depth value at image coordinate
        const float md   = depth.GetBilinear<float>(p_c);

        // normal value at image coordinate
        const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

        const float costheta = dot(mdn, P_c) / -length(P_c);
        const float sd = costheta * (md - vd);
        const float w = costheta * 1.0f/vd;

        if(sd <= -trunc_dist)
        {
          // Further than truncation distance from surface
          // We do nothing.
        }
        // update SDF
        else
        {
          //        }else if(sd < 5*trunc_dist) {

          /// here 0.5 is for kinect sensor
          int nIndex = g_vol.GetIndex(int(floorf(x/g_vol.m_nVolumeGridRes)),
                                      int(floorf(y/g_vol.m_nVolumeGridRes)),
                                      int(floorf(z/g_vol.m_nVolumeGridRes)) );

          if(/*sd < 5*trunc_dist && */isfinite(md) && md>0.5 && costheta > mincostheta )
          {
            if(g_vol.CheckIfBasicSDFActive(nIndex)==true )
            {
              const SDF_t curvol = g_vol(x,y,z);

              // return min of 'sd' and 'trunc_dist' as 'x', then rerurn max of 'x' and 'w'
              SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
              sdf += curvol;
              sdf.LimitWeight(max_w);

              /// set val
              g_vol(x, y, z) = sdf;
              g_grayVol(x,y,z) = (w*c + g_grayVol(x,y,z) * curvol.w) / (w + curvol.w);
//              printf("fuse:%f,", c);
            }
            else
            {
              printf("[KernSdfFuseDirectgrayGridSafe] warnning!!! skip %d,%d,%d when fusing!!!\n",
                     int(floorf(x/g_vol.m_nVolumeGridRes)),
                     int(floorf(y/g_vol.m_nVolumeGridRes)),
                     int(floorf(z/g_vol.m_nVolumeGridRes)) );
            }

          }
        }
      }
    }
  }
}


void SdfFuseDirectGrayGridSafe(
    BoundedVolumeGrid<SDF_t, roo::TargetDevice, roo::Manage> vol,
    BoundedVolumeGrid<float, roo::TargetDevice, roo::Manage> colorVol,
    Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
    Image<float> gray, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
    float trunc_dist, float max_w, float mincostheta
    )
{
  /// load grid sdf to golbal memory. We do this because there is a size limit of
  // the parameters that we can send the the kernel function.
  hipMemcpyToSymbol(HIP_SYMBOL(g_vol), &vol, sizeof(vol), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(g_grayVol), &colorVol, sizeof(colorVol), size_t(0), hipMemcpyHostToDevice);
  GpuCheckErrors();

  // launch kernel for SDF fusion
  dim3 blockDim(32,32);
  dim3 gridDim(vol.m_w / blockDim.x, vol.m_h / blockDim.y);
  KernSdfFuseDirectGrayGridSafe<<<gridDim,blockDim>>>(depth, norm, T_cw, Kdepth, gray, T_iw, Krgb, trunc_dist, max_w, mincostheta);
  GpuCheckErrors();

  // copy data back after launch the kernel
  vol.CopyFrom(g_vol);
  colorVol.CopyFrom(g_grayVol);
  GpuCheckErrors();

  // cuda free memory
  g_vol.FreeMemory();
  g_grayVol.FreeMemory();
  GpuCheckErrors();
}



// -----------------------------------------------------------------------------
//--the following add by luma---------------------------------------------------
// do SDF fusion for certain index without consideing void (zero intensity) pixels
// notice that in this function, we check each voxel we have and see if we need
// to fuse any information into it. This is different from check each pixel we have
// and try to fuse it into the sdf.
__global__ void KernSdfFuseDirectgrayGridDesireIndex(
    Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
    Image<float> gray, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
    float trunc_dist, float max_w, float mincostheta, bool bWeight
    )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  //    const int z = blockIdx.z*blockDim.z + threadIdx.z;

  // For each voxel (x,y,z) we have in a bounded volume
  for(int z=0; z < g_vol.m_d; ++z)
  {
    int nIndex = g_vol.GetIndex(int(floorf(x/g_vol.m_nVolumeGridRes)),
                                int(floorf(y/g_vol.m_nVolumeGridRes)),
                                int(floorf(z/g_vol.m_nVolumeGridRes)) );

    if(g_NextInitSDFs[nIndex] == 1)
    {
      // See if this voxel is possible to be in the image boundary
      // Get voxel position in certain radius in world coordinate (good)
      const float3 P_w = g_vol.VoxelPositionInUnits(x,y,z);

      // Get voxel position in camera coordinate (good)
      const float3 P_c = T_cw * P_w;

      // Project a 3D voxel point to 2D depth an gray image coordinate
      const float2 p_c = Kdepth.Project(P_c);

      const float3 P_i = T_iw * P_w;
      const float2 p_i = Krgb.Project(P_i);

      // If the voxel is in image coordinate (inside of image boundary), then we
      // see if we should fuse this voxel
      if( depth.InBounds(p_c, 2) && gray.InBounds(p_i,2) )
      {
        // prepare to fuse a gray pixel into this voxel
        const float c =  gray.GetBilinear<float>(p_i);

        // discard pixel value equals 0
        if(c!=0)
        {
          // depth value at camera coorniate
          const float vd   = P_c.z;

          // depth value at image coordinate
          const float md   = depth.GetBilinear<float>(p_c);

          // normal value at image coordinate
          const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

          const float costheta = dot(mdn, P_c) / -length(P_c);
          const float sd = costheta * (md - vd);
          const float w = costheta * 1.0f/vd;

          if(sd <= -trunc_dist)
          {
            // Further than truncation distance from surface
            // We do nothing.
          }
          // update SDF
          else
          {
            //        }else if(sd < 5*trunc_dist) {

            /// here 0.5 is for kinect sensor
            if(/*sd < 5*trunc_dist && */isfinite(md) && md>0.5 && costheta > mincostheta )
            {
              const SDF_t curvol = g_vol(x,y,z);

              // return min of 'sd' and 'trunc_dist' as 'x', then rerurn max of 'x' and 'w'
              SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
              sdf += curvol;
              sdf.LimitWeight(max_w);

              // set val
              g_vol(x, y, z) = sdf;

              if(bWeight == true)
              {
                g_grayVol(x,y,z) = (w*c + g_grayVol(x,y,z) * curvol.w) / (w + curvol.w);
              }
              else
              {
                if(g_grayVol(x,y,z)>0)
                {
                  printf("skipFuse;");
                }
                else
                {
                  g_grayVol(x,y,z) = (w*c + g_grayVol(x,y,z) * curvol.w) / (w + curvol.w);
                }
              }

            }
          }
        }
      }
    }
  }
}


void SdfFuseDirectGrayGridDesireIndex(
    int* pNextInitSDFs,
    BoundedVolumeGrid<SDF_t, roo::TargetDevice, roo::Manage> vol,
    BoundedVolumeGrid<float, roo::TargetDevice, roo::Manage> colorVol,
    Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
    Image<float> gray, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
    float trunc_dist, float max_w, float mincostheta, bool bWeight
    )
{
  if(vol.m_nWholeGridRes*vol.m_nWholeGridRes*vol.m_nWholeGridRes>102400)
  {
    printf("[SdfFuseDirectgrayGridAutoInit] Fatal Error! Array size overflow!\n");
    exit(-1);
  }

  /// load grid sdf to golbal memory. We do this because there is a size limit of
  // the parameters that we can send the the kernel function.
  hipMemcpyToSymbol(HIP_SYMBOL(g_vol), &vol, sizeof(vol), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(g_grayVol), &colorVol, sizeof(colorVol), size_t(0), hipMemcpyHostToDevice);
  GpuCheckErrors();

  // copy array back
  int nNextInitSDFs[102400];

  for(int i=0;i!=vol.m_nWholeGridRes*vol.m_nWholeGridRes*vol.m_nWholeGridRes;i++)
  {
    nNextInitSDFs[i] = pNextInitSDFs[i] ;
  }

  hipMemcpyToSymbol(HIP_SYMBOL(g_NextInitSDFs), nNextInitSDFs, sizeof(nNextInitSDFs), 0, hipMemcpyHostToDevice);
  GpuCheckErrors();

  // launch kernel for SDF fusion
  dim3 blockDim(32,32);
  dim3 gridDim(vol.m_w / blockDim.x, vol.m_h / blockDim.y);
  KernSdfFuseDirectgrayGridDesireIndex<<<gridDim,blockDim>>>(depth, norm, T_cw, Kdepth, gray, T_iw, Krgb, trunc_dist, max_w, mincostheta, bWeight);
  GpuCheckErrors();

  // copy data back after launch the kernel
  vol.CopyFrom(g_vol);
  colorVol.CopyFrom(g_grayVol);
  GpuCheckErrors();

  // cuda free memory
  g_vol.FreeMemory();
  g_grayVol.FreeMemory();
  GpuCheckErrors();

  printf("[SdfFuseDirectgrayGridDesireIndex/cu] Finished all.\n");
}



// -----------------------------------------------------------------------------
//--the following add by luma---------------------------------------------------
// the following do Grid SDF fusion and also mark voxels that cannot be fused in
// due to uninitialized Grid SDF.
__global__ void KernSdfFuseDirectGrayGridAutoInit(
    Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
    Image<float> gray, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
    float trunc_dist, float max_w, float mincostheta, bool bWeight
    )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  //    const int z = blockIdx.z*blockDim.z + threadIdx.z;

  // For each voxel (x,y,z) we have in a bounded volume
  for(int z=0; z < g_vol.m_d; ++z)
  {
    // See if this voxel is possible to be in the image boundary
    // Get voxel position in certain radius in world coordinate (good)
    const float3 P_w = g_vol.VoxelPositionInUnits(x,y,z);

    // Get voxel position in camera coordinate (good)
    const float3 P_c = T_cw * P_w;

    // Project a 3D voxel point to 2D depth an gray image coordinate
    const float2 p_c = Kdepth.Project(P_c);

    const float3 P_i = T_iw * P_w;
    const float2 p_i = Krgb.Project(P_i);

    // If the voxel is in image coordinate (inside of image boundary), then we
    // see if we should fuse this voxel
    if( depth.InBounds(p_c, 2) && gray.InBounds(p_i,2) )
    {
      // prepare to fuse a gray pixel into this voxel
      const float c =  gray.GetBilinear<float>(p_i);

      // discard pixel value equals 0
      if(c!=0)
      {
        // depth value at camera coorniate
        const float vd   = P_c.z;

        // depth value at image coordinate
        const float md   = depth.GetBilinear<float>(p_c);

        // normal value at image coordinate
        const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

        const float costheta = dot(mdn, P_c) / -length(P_c);
        const float sd = costheta * (md - vd);
        const float w = costheta * 1.0f/vd;

        if(sd <= -trunc_dist)
        {
          // Further than truncation distance from surface
          // We do nothing.
        }
        // update SDF
        else
        {
          //        }else if(sd < 5*trunc_dist) {

          /// here 0.5 is for kinect sensor
          if(/*sd < 5*trunc_dist && */isfinite(md) && md>0.5 && costheta > mincostheta )
          {
            int nIndex = g_vol.GetIndex(int(floorf(x/g_vol.m_nVolumeGridRes)),
                                        int(floorf(y/g_vol.m_nVolumeGridRes)),
                                        int(floorf(z/g_vol.m_nVolumeGridRes)) );

            if(g_vol.CheckIfBasicSDFActive(nIndex) == true)
            {
              const SDF_t curvol = g_vol(x,y,z);

              // return min of 'sd' and 'trunc_dist' as 'x', then rerurn max of 'x' and 'w'
              SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
              sdf += curvol;
              sdf.LimitWeight(max_w);

              /// set val
              g_vol(x, y, z) = sdf;

              if(bWeight == true)
              {
                g_grayVol(x,y,z) = (w*c + g_grayVol(x,y,z) * curvol.w) / (w + curvol.w);
              }
              else
              {
                if(g_grayVol(x,y,z)==0.5)
                {
                  g_grayVol(x,y,z) = (w*c + g_grayVol(x,y,z) * curvol.w) / (w + curvol.w);
                }
              }

            }
            else
            {
              g_NextInitSDFs[nIndex] = 1;
            }

          }
        }
      }
    }
  }
}


void SdfFuseDirectGrayGridAutoInit(
    int* pNextInitSDFs,
    BoundedVolumeGrid<SDF_t, roo::TargetDevice, roo::Manage> vol,
    BoundedVolumeGrid<float, roo::TargetDevice, roo::Manage> colorVol,
    Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
    Image<float> gray, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
    float trunc_dist, float max_w, float mincostheta, bool bWeight
    )
{
  if(vol.m_nWholeGridRes*vol.m_nWholeGridRes*vol.m_nWholeGridRes>102400)
  {
    printf("[SdfFuseDirectgrayGridAutoInit] Fatal Error! Array size overflow!\n");
    exit(-1);
  }

  /// load grid sdf to golbal memory. We do this because there is a size limit of
  // the parameters that we can send the the kernel function.
  hipMemcpyToSymbol(HIP_SYMBOL(g_vol), &vol, sizeof(vol), size_t(0), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(g_grayVol), &colorVol, sizeof(colorVol), size_t(0), hipMemcpyHostToDevice);
  GpuCheckErrors();

  // launch kernel for SDF fusion
  dim3 blockDim(32,32);
  dim3 gridDim(vol.m_w / blockDim.x, vol.m_h / blockDim.y);
  KernSdfFuseDirectGrayGridAutoInit<<<gridDim,blockDim>>>(depth, norm, T_cw, Kdepth, gray, T_iw, Krgb, trunc_dist, max_w, mincostheta, bWeight);
  GpuCheckErrors();

  // check if need to init new grid sdf
  int nNextInitSDFs[102400];
  hipMemcpyFromSymbol(nNextInitSDFs, HIP_SYMBOL(g_NextInitSDFs), sizeof(g_NextInitSDFs), 0, hipMemcpyDeviceToHost);
  GpuCheckErrors();

  // copy array back
  for(int i=0;i!=vol.m_nWholeGridRes*vol.m_nWholeGridRes*vol.m_nWholeGridRes;i++)
  {
    pNextInitSDFs[i] = nNextInitSDFs[i];
    nNextInitSDFs[i] = 0;
  }

  // reset index
  hipMemcpyToSymbol(HIP_SYMBOL(g_NextInitSDFs),nNextInitSDFs,sizeof(nNextInitSDFs),0,hipMemcpyHostToDevice);
  GpuCheckErrors();

  // copy data back after launch the kernel
  vol.CopyFrom(g_vol);
  colorVol.CopyFrom(g_grayVol);
  GpuCheckErrors();

  // cuda free memory
  g_vol.FreeMemory();
  g_grayVol.FreeMemory();
  GpuCheckErrors();
}


// -----------------------------------------------------------------------------
//--the following add by luma---------------------------------------------------
// do SDF fusion without consideing void (zero intensity) pixels
//__global__ void KernSdfFuseDirectgrayGrid(float* depth, float4* normals, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
//                                          Image<float> gray, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
//                                          float trunc_dist, float max_w, float mincostheta
//                                          )
//{
//  const int x = blockIdx.x*blockDim.x + threadIdx.x;
//  const int y = blockIdx.y*blockDim.y + threadIdx.y;

//  //    const int z = blockIdx.z*blockDim.z + threadIdx.z;

//  // For each voxel (x,y,z) we have in a bounded volume
//  for(int z=0; z < g_vol.m_d; ++z)
//  {
//    // See if this voxel is possible to be in the image boundary
//    // Get voxel position in certain radius in world coordinate (good)
//    const float3 P_w = g_vol.VoxelPositionInUnits(x,y,z);

//    // Get voxel position in camera coordinate (good)
//    const float3 P_c = T_cw * P_w;

//    // Project a 3D voxel point to 2D depth an gray image coordinate
//    const float2 p_c = Kdepth.Project(P_c);

//    const float3 P_i = T_iw * P_w;
//    const float2 p_i = Krgb.Project(P_i);

//    // If the voxel is in image coordinate (inside of image boundary), then we
//    // see if we should fuse this voxel
//    if( depth.InBounds(p_c, 2) && gray.InBounds(p_i,2) )
//    {
//      // prepare to fuse a gray pixel into this voxel
//      const float c =  gray.GetBilinear<float>(p_i);

//      // discard pixel value equals 0
//      if(c!=0)
//      {
//        // depth value at camera coorniate
//        const float vd   = P_c.z;

//        // depth value at image coordinate
//        const float md   = depth.GetBilinear<float>(p_c);

//        // normal value at image coordinate
//        const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

//        const float costheta = dot(mdn, P_c) / -length(P_c);
//        const float sd = costheta * (md - vd);
//        const float w = costheta * 1.0f/vd;

//        if(sd <= -trunc_dist)
//        {
//          // Further than truncation distance from surface
//          // We do nothing.
//        }
//        // update SDF
//        else
//        {
//          //        }else if(sd < 5*trunc_dist) {

//          /// here 0.5 is for kinect sensor
//          if(/*sd < 5*trunc_dist && */isfinite(md) && md>0.5 && costheta > mincostheta )
//          {
//            //            printf("fuse:x%d,y%d,z%d",x,y,z);

////            printf("shift:%d",g_vol.m_shift.x);

//            const SDF_t curvol = g_vol(x,y,z);

//            // return min of 'sd' and 'trunc_dist' as 'x', then rerurn max of 'x' and 'w'
//            SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
//            sdf += curvol;
//            sdf.LimitWeight(max_w);

//            /// set val
//            g_vol(x, y, z) = sdf;
//            g_colorVol(x,y,z) = (w*c + g_colorVol(x,y,z) * curvol.w) / (w + curvol.w);
//          }
//        }
//      }
//    }
//  }
//}


//void SdfFuseDirectgrayGrid(
//    BoundedVolumeGrid<SDF_t, roo::TargetDevice, roo::Manage> vol,
//    BoundedVolumeGrid<float, roo::TargetDevice, roo::Manage> colorVol,
//    float* depth, float4* norm, Mat<float,3,4> T_cw, ImageIntrinsics Kdepth,
//    Image<float> gray, Mat<float,3,4> T_iw, ImageIntrinsics Krgb,
//    float trunc_dist, float max_w, float mincostheta
//    )
//{
//  /// load grid sdf to golbal memory. We do this because there is a size limit of
//  // the parameters that we can send the the kernel function.
//  hipMemcpyToSymbol(HIP_SYMBOL(g_vol), &vol, sizeof(vol), size_t(0), hipMemcpyHostToDevice);
//  hipMemcpyToSymbol(HIP_SYMBOL(g_colorVol), &colorVol, sizeof(colorVol), size_t(0), hipMemcpyHostToDevice);
//  GpuCheckErrors();

//  // launch kernel for SDF fusion
//  dim3 blockDim(32,32);
//  dim3 gridDim(vol.m_w / blockDim.x, vol.m_h / blockDim.y);
//  KernSdfFuseDirectgrayGrid<<<gridDim,blockDim>>>(depth, norm, T_cw, Kdepth, gray, T_iw, Krgb, trunc_dist, max_w, mincostheta);
//  GpuCheckErrors();

//  // copy data back after launch the kernel
//  vol.CopyFrom(g_vol);
//  colorVol.CopyFrom(g_colorVol);
//  GpuCheckErrors();

//  // cuda free memory
//  g_vol.FreeMemory();
//  g_colorVol.FreeMemory();
//  GpuCheckErrors();
//}


// -----------------------------------------------------------------------------
//--the following add by luma---------------------------------------------------
// do SDF fusion without consideing void (zero intensity) pixels
__global__ void KernSdfFuseColor(
    BoundedVolume<SDF_t> vol, BoundedVolume<uchar3> colorVol,
    Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics K,
    Image<float> img, Image<uchar3> Imgrgb, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
    float trunc_dist, float max_w, float mincostheta
    )
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  //    const int z = blockIdx.z*blockDim.z + threadIdx.z;
  for(int z=0; z < vol.d; ++z)
  {
    const float3 P_w = vol.VoxelPositionInUnits(x,y,z);
    const float3 P_c = T_cw * P_w;
    const float2 p_c = K.Project(P_c);
    const float3 P_i = T_iw * P_w;
    const float2 p_i = Kimg.Project(P_i);

    if( depth.InBounds(p_c, 2) && img.InBounds(p_i,2) )
    {
      const float c =  img.GetBilinear<float>(p_i);

      // discard pixel value equals 0
      if(c!=0)
      {
        const float vd = P_c.z;
        const float md = depth.GetBilinear<float>(p_c);
        const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

        const float costheta = dot(mdn, P_c) / -length(P_c);
        const float sd = costheta * (md - vd);
        //                const float w = 1;
        const float w = costheta * 1.0f/vd;

        if(sd <= -trunc_dist)
        {
          // Further than truncation distance from surface
          // We do nothing.
        }
        // update SDF
        else
        {
          //        }else if(sd < 5*trunc_dist) {
          if(/*sd < 5*trunc_dist && */isfinite(md) && md>0.5 && costheta > mincostheta )
          {
            const SDF_t curvol = vol(x,y,z);
            // return min of 'sd' and 'trunc_dist' as 'x', then rerurn max of 'x' and 'w'
            SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
            sdf += curvol;
            sdf.LimitWeight(max_w);
            vol(x,y,z) = sdf;

            //            printf("(u,v)=(%d,%d),(r,g,b)=(%d,%d,%d),(x,y,z)=(%d,%d,%d)",int(p_i.x),int(p_i.y),
            //                   int(Imgrgb.Get(int(p_i.x),int(p_i.y)).x),
            //                   int(Imgrgb.Get(int(p_i.x),int(p_i.y)).y),
            //                   int(Imgrgb.Get(int(p_i.x),int(p_i.y)).z),
            //                   x,y,z);

            colorVol(x,y,z) = make_uchar3( Imgrgb.Get(int(p_i.x),int(p_i.y)).x,
                                           Imgrgb.Get(int(p_i.x),int(p_i.y)).y,
                                           Imgrgb.Get(int(p_i.x),int(p_i.y)).z) ;
          }
        }
      }
    }
  }
}

void SdfFuseColor(
    BoundedVolume<SDF_t> vol, BoundedVolume<uchar3> colorVol,
    Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics K,
    Image<float> img, Image<uchar3> Imgrgb, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
    float trunc_dist, float max_w, float mincostheta
    ) {
  dim3 blockDim(16,16);
  dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y);
  KernSdfFuseColor<<<gridDim,blockDim>>>(vol, colorVol, depth, norm,
                                         T_cw, K, img, Imgrgb, T_iw, Kimg,
                                         trunc_dist, max_w, mincostheta);
  GpuCheckErrors();
}


//////////////////////////////////////////////////////
// Reset SDF
//////////////////////////////////////////////////////



void SdfReset(BoundedVolume<SDF_t> vol, float trunc_dist)
{
  vol.Fill(SDF_t(0.0/0.0, 0));
}

void SdfReset(BoundedVolume<float> vol)
{
  vol.Fill(0.5);
}

void SdfReset(BoundedVolume<uchar3> vol)
{
  vol.Fill(make_uchar3( 0,0,0 ));
}

void SdfReset(VolumeGrid<SDF_t,roo::TargetDevice, roo::Manage> vol)
{
  vol.Fill(SDF_t(0.0/0.0, 0));
}

void SdfReset(VolumeGrid<float,roo::TargetDevice, roo::Manage> vol)
{
  vol.Fill(0.5);
}

void SdfReset(BoundedVolumeGrid<float,roo::TargetDevice, roo::Manage> vol)
{
  for(unsigned int i=0;i!=vol.m_nWholeGridRes*vol.m_nWholeGridRes*vol.m_nWholeGridRes;i++)
  {
    // reset for each valid rolling grid sdf
    if(vol.CheckIfBasicSDFActive(i)==true)
    {
      roo::SdfReset(vol.m_GridVolumes[i]);
    }
  }
}


void SdfReset(BoundedVolumeGrid<roo::SDF_t,roo::TargetDevice, roo::Manage> vol)
{
  for(unsigned int i=0;i!=vol.m_nWholeGridRes*vol.m_nWholeGridRes*vol.m_nWholeGridRes;i++)
  {
    // reset for each valid rolling grid sdf
    if(vol.CheckIfBasicSDFActive(i)==true)
    {
      roo::SdfReset(vol.m_GridVolumes[i]);
    }
  }
}


//////////////////////////////////////////////////////
// Create SDF representation of sphere
//////////////////////////////////////////////////////

__global__ void KernSdfSphere(BoundedVolume<SDF_t> vol, float3 center, float r)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  const int z = blockIdx.z*blockDim.z + threadIdx.z;

  const float3 pos = vol.VoxelPositionInUnits(x,y,z);
  const float dist = length(pos - center);
  const float sdf = dist - r;

  vol(x,y,z) = SDF_t(sdf);
}

void SdfSphere(BoundedVolume<SDF_t> vol, float3 center, float r)
{
  dim3 blockDim(8,8,8);
  dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);

  KernSdfSphere<<<gridDim,blockDim>>>(vol, center, r);
  GpuCheckErrors();
}

//////////////////////////////////////////////////////
// Take SDF Difference to depthmap
//////////////////////////////////////////////////////

__global__ void KernSdfDistance(Image<float> dist, Image<float> depth, BoundedVolume<SDF_t> vol, const Mat<float,3,4> T_wc, ImageIntrinsics K, float trunc_distance)
{
  const int u = blockIdx.x*blockDim.x + threadIdx.x;
  const int v = blockIdx.y*blockDim.y + threadIdx.y;

  if( u < depth.w && v < depth.h ) {
    const float z = depth(u,v);
    const float3 p_c = z * K.Unproject(u,v);
    const float3 p_w = T_wc * p_c;

    const SDF_t sdf = vol.GetUnitsTrilinearClamped(p_w);
    dist(u,v) = sdf.val; //(sdf.val + trunc_distance) / (2* trunc_distance);
  }
}


void SdfDistance(Image<float> dist, Image<float> depth, BoundedVolume<SDF_t> vol, const Mat<float,3,4> T_wc, ImageIntrinsics K, float trunc_distance)
{
  dim3 blockDim, gridDim;
  InitDimFromOutputImageOver(blockDim, gridDim, depth);

  KernSdfDistance<<<gridDim,blockDim>>>(dist, depth, vol, T_wc, K, trunc_distance);
  GpuCheckErrors();
}

}
