#include "hip/hip_runtime.h"
#include "cu_sdffusion.h"

#include "MatUtils.h"
#include "launch_utils.h"

namespace roo
{

//////////////////////////////////////////////////////
// Truncated SDF Fusion
// KinectFusion: Real-Time Dense Surface Mapping and Tracking, Newcombe et. al.
// http://www.doc.ic.ac.uk/~rnewcomb/
//////////////////////////////////////////////////////

__global__ void KernSdfFuse(BoundedVolume<SDF_t> vol, Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics K, float trunc_dist, float max_w, float mincostheta )
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    const int z = blockIdx.z*blockDim.z + threadIdx.z;

    const float3 P_w = vol.VoxelPositionInUnits(x,y,z);
    const float3 P_c = T_cw * P_w;
    const float2 p_c = K.Project(P_c);

    if( depth.InBounds(p_c, 2) )
    {
        const float vd = P_c.z;
//        const float md = depth.GetNearestNeighbour(p_c);
//        const float3 mdn = make_float3(normals.GetNearestNeighbour(p_c));

        const float md = depth.GetBilinear<float>(p_c);
        const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

        const float costheta = dot(mdn, P_c) / -length(P_c);
        const float sd = costheta * (md - vd);
        const float w = costheta * 1.0f/vd;

        if(sd <= -trunc_dist) {
            // Further than truncation distance from surface
            // We do nothing.
        }else{
//        }else if(sd < 5*trunc_dist) {
            if(isfinite(md) && isfinite(w) && costheta > mincostheta ) {
                SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
                sdf += vol(x,y,z);
//                sdf.Clamp(-trunc_dist, trunc_dist);
                sdf.LimitWeight(max_w);
                vol(x,y,z) = sdf;
            }
        }
    }
 }

void SdfFuse(BoundedVolume<SDF_t> vol, Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics K, float trunc_dist, float max_w, float mincostheta)
{
    dim3 blockDim(8,8,8);
    dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);
    KernSdfFuse<<<gridDim,blockDim>>>(vol, depth, norm, T_cw, K, trunc_dist, max_w, mincostheta);
    GpuCheckErrors();
}

//////////////////////////////////////////////////////
// Color Truncated SDF Fusion
// Similar extension to KinectFusion as described by:
// Robust Tracking for Real-Time Dense RGB-D Mapping with Kintinous
// Whelan et. al.
//////////////////////////////////////////////////////

__global__ void KernSdfFuse(
        BoundedVolume<SDF_t> vol, BoundedVolume<float> colorVol,
        Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics K,
        Image<uchar3> img, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
        float trunc_dist, float max_w, float mincostheta
        )
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

//    const int z = blockIdx.z*blockDim.z + threadIdx.z;
    for(int z=0; z < vol.d; ++z) {
        const float3 P_w = vol.VoxelPositionInUnits(x,y,z);
        const float3 P_c = T_cw * P_w;
        const float2 p_c = K.Project(P_c);
        const float3 P_i = T_iw * P_w;
        const float2 p_i = Kimg.Project(P_i);

        if( depth.InBounds(p_c, 2) && img.InBounds(p_i,2) )
        {
            const float vd = P_c.z;
//            const float md = depth.GetNearestNeighbour(p_c);
//            const float3 mdn = make_float3(normals.GetNearestNeighbour(p_c));
//            const float c = ConvertPixel<float,uchar3>( img.GetNearestNeighbour(p_i) );

            const float md = depth.GetBilinear<float>(p_c);
            const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));
            const float c = ConvertPixel<float,float3>( img.GetBilinear<float3>(p_i) ) / 255.0;

            const float costheta = dot(mdn, P_c) / -length(P_c);
            const float sd = costheta * (md - vd);
            const float w = costheta * 1.0f/vd;

            if(sd <= -trunc_dist) {
                // Further than truncation distance from surface
                // We do nothing.
            }else{
    //        }else if(sd < 5*trunc_dist) {
                if(isfinite(md) && isfinite(w) && costheta > mincostheta ) {
                    const SDF_t curvol = vol(x,y,z);
                    SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
                    sdf += curvol;
                    sdf.LimitWeight(max_w);
                    vol(x,y,z) = sdf;
                    colorVol(x,y,z) = (w*c + colorVol(x,y,z) * curvol.w) / (w + curvol.w);
                }
            }
        }
    }
 }

void SdfFuse(
        BoundedVolume<SDF_t> vol, BoundedVolume<float> colorVol,
        Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics K,
        Image<uchar3> img, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
        float trunc_dist, float max_w, float mincostheta
) {
//    // 3d invoke
//    dim3 blockDim(8,8,8);
//    dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);
//    KernSdfFuse<<<gridDim,blockDim>>>(vol, colorVol, depth, norm, T_cw, K, img, T_iw, Kimg, trunc_dist, max_w, mincostheta);
//    GpuCheckErrors();

    dim3 blockDim(16,16);
    dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y);
    KernSdfFuse<<<gridDim,blockDim>>>(vol, colorVol, depth, norm, T_cw, K, img, T_iw, Kimg, trunc_dist, max_w, mincostheta);
    GpuCheckErrors();

}


//--the following add by luma-----------------------------------------------------------------------------------------------------------------------------

__global__ void KernSdfFuseDirectGrey(
        BoundedVolume<SDF_t> vol, BoundedVolume<float> colorVol,
        Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics K,
        Image<float> img, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
        float trunc_dist, float max_w, float mincostheta
        )
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

//    const int z = blockIdx.z*blockDim.z + threadIdx.z;
    for(int z=0; z < vol.d; ++z)
    {
        const float3 P_w = vol.VoxelPositionInUnits(x,y,z);
        const float3 P_c = T_cw * P_w;
        const float2 p_c = K.Project(P_c);
        const float3 P_i = T_iw * P_w;
        const float2 p_i = Kimg.Project(P_i);

        if( depth.InBounds(p_c, 2) && img.InBounds(p_i,2) )
        {
            const float c =  img.GetBilinear<float>(p_i);

            // discard pixel value equals 0
            if(c!=0)
            {
                const float vd = P_c.z;
                const float md = depth.GetBilinear<float>(p_c);
                const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));

                const float costheta = dot(mdn, P_c) / -length(P_c);
                const float sd = costheta * (md - vd);
//                const float w = 1;
                const float w = costheta * 1.0f/vd;

                if(sd <= -trunc_dist)
                {
                    // Further than truncation distance from surface
                    // We do nothing.
                }
                else
                {
        //        }else if(sd < 5*trunc_dist) {
                    if(/*sd < 5*trunc_dist && */isfinite(md) && md!=0 && costheta > mincostheta )
                    {
                        const SDF_t curvol = vol(x,y,z);
                        SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w); // return min of 'sd' and 'trunc_dist' as 'x', then rerurn max of 'x' and 'w'
                        sdf += curvol;
                        sdf.LimitWeight(max_w);
                        vol(x,y,z) = sdf;
                        colorVol(x,y,z) = (w*c + colorVol(x,y,z) * curvol.w) / (w + curvol.w);
                    }
                }
            }
        }
    }
 }

void SdfFuseDirectGrey(
        BoundedVolume<SDF_t> vol, BoundedVolume<float> colorVol,
        Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics K,
        Image<float> img, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
        float trunc_dist, float max_w, float mincostheta
) {
    dim3 blockDim(16,16);
    dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y);
    KernSdfFuseDirectGrey<<<gridDim,blockDim>>>(vol, colorVol, depth, norm, T_cw, K, img, T_iw, Kimg, trunc_dist, max_w, mincostheta);
    GpuCheckErrors();

}


//-----------------------------------------------------------------------------------------------------------------------------------

//////////////////////////////////////////////////////
// find outline from sdf
/////////////////////////////////////////////////////
__global__ void KernSdfFuseFindOutline(
        BoundedVolume<SDF_t> vol,BoundedVolume<float> colorVol,
        Image<float> depth, Image<float4> normals, Mat<float,3,4> T_cw, ImageIntrinsics K,
        Image<uchar3> img, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
        float trunc_dist, float max_w, float mincostheta, Image<float4> dOutLine
        )
{

    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
//    const int z = blockIdx.z*blockDim.z + threadIdx.z;


    for(int z=0; z < vol.d; ++z)
    {
        const float3 P_w = vol.VoxelPositionInUnits(x,y,z);
        const float3 P_c = T_cw * P_w;
        const float2 p_c = K.Project(P_c);
        const float3 P_i = T_iw * P_w;
        const float2 p_i = Kimg.Project(P_i);

        // if voxel is inside bounds
        if( depth.InBounds(p_c, 2) && img.InBounds(p_i,2) )
        {
            const float vd = P_c.z;

            const float md = depth.GetBilinear<float>(p_c);
            const float3 mdn = make_float3(normals.GetBilinear<float4>(p_c));
            const float c = ConvertPixel<float,float3>( img.GetBilinear<float3>(p_i) ) / 255.0;

            const float costheta = dot(mdn, P_c) / -length(P_c);
            const float sd = costheta * (md - vd);
            const float w = costheta * 1.0f/vd;

            if(sd <= -trunc_dist)
            {
            }
            else
            {
                SDF_t sdf( clamp(sd,-trunc_dist,trunc_dist) , w);
                sdf += vol(x,y,z);

                sdf.LimitWeight(max_w);
                vol(x,y,z) = sdf;
            }
        }
        // out of boundary
        else
        {
//            dOutLine(p_i.x,p_i.y) = make_float4(1, 0, 1, 1);
        }
    }

 }

void SdfFuseFindOutline(
        BoundedVolume<SDF_t> vol, BoundedVolume<float> colorVol,Image<float> depth, Image<float4> norm, Mat<float,3,4> T_cw, ImageIntrinsics K,
        Image<uchar3> img, Mat<float,3,4> T_iw, ImageIntrinsics Kimg,
        float trunc_dist, float max_w, float mincostheta, Image<float4> dOutline
) {

    dim3 blockDim(16,16);
    dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y);
    KernSdfFuseFindOutline<<<gridDim,blockDim>>>(vol, colorVol, depth, norm, T_cw, K, img, T_iw, Kimg, trunc_dist, max_w, mincostheta, dOutline);
    GpuCheckErrors();
}


__global__ void KernFindBBBoundary(
       BoundedVolume<float> colorVol, int max_x, int max_y, int max_z, int min_x, int min_y, int min_z
        )
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;

//    const int z = blockIdx.z*blockDim.z + threadIdx.z;
    for(int z=0; z < colorVol.d; ++z)
    {
        const float curvol = colorVol(x,y,z);

        if(isfinite(curvol))
        {
//            printf(",vol:%f", curvol);
            if(x>max_x)
            {
                max_x = x;
            }
            else if(x<min_x)
            {
                min_x = x;
            }

            if(y>max_y)
            {
                max_y = y;
            }
            else if(y<min_y)
            {
                min_y = y;
            }

            if(z>max_z)
            {
                max_z = z;
            }
            else if(z<min_z)
            {
                min_z = z;
            }
        }
    }
 }

void FindBBBoundary(
        BoundedVolume<float> colorVol, int max_x, int max_y, int max_z, int min_x, int min_y, int min_z
) {
    dim3 blockDim(16,16);
    dim3 gridDim(colorVol.w / blockDim.x, colorVol.h / blockDim.y);
    KernFindBBBoundary<<<gridDim,blockDim>>>(colorVol, max_x, max_y, max_z, min_x, min_y, min_z);
    GpuCheckErrors();

}

// ---------------------------------------------------------------------------------------------------------------------------------


//////////////////////////////////////////////////////
// Reset SDF
//////////////////////////////////////////////////////

void SdfReset(BoundedVolume<SDF_t> vol, float trunc_dist)
{
    vol.Fill(SDF_t(0.0/0.0, 0));
}

void SdfReset(BoundedVolume<float> vol)
{
    vol.Fill(0.5);
}


//boxmin and boxmax define the box that is to be kept intact, rest will be cleared. This approach makes if conditions inside simpler.
//TODO: Name the function better.
__global__ void KernSdfResetPartial(BoundedVolume<SDF_t> vol, float3 boxmin, float3 boxmax)
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    const int z = blockIdx.z*blockDim.z + threadIdx.z;

    const float3 P_w = vol.VoxelPositionInUnits(x,y,z);

    bool mincrit, maxcrit;//if mincrit and maxcrit are true, point is inside the box, i.e. valid.
    mincrit = P_w.x > boxmin.x && P_w.y < boxmax.y && P_w.z > boxmin.z;
    maxcrit = P_w.x < boxmax.x && P_w.y > boxmin.y && P_w.z < boxmax.z;

    if(!mincrit || !maxcrit)//i.e. the point is outside the box.
    {
    vol(x,y,z) = SDF_t(0.0/0.0,0.0);
    }
}

//TODO: Name the function better.
void SdfResetPartial(BoundedVolume<SDF_t> vol, float3 shift)
{
    //Initialization for GPU parallelization
    dim3 blockDim(8,8,8);
    dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);

    //compute the box to keep, it's conter intuitive to the name of function but more efficient.
    float3 bn = vol.bbox.boxmin, bx = vol.bbox.boxmax;//bn is box min and bx is box max.

    if(shift.x>0)
    bn.x += shift.x;
    else
    bx.x += shift.x;

    //y is -ve, but boxmax and boxmin for y are also inverse. i.e. the bottom most point is min.x,max.y,min.z
    if(shift.y>0)
    bn.y += shift.y;
    else
    bx.y += shift.y;

    if(shift.z>0)
    bn.z += shift.z;
    else
    bx.z += shift.z;

    KernSdfResetPartial<<<gridDim,blockDim>>>(vol, bn, bx);
    GpuCheckErrors();

}



//////////////////////////////////////////////////////
// Create SDF representation of sphere
//////////////////////////////////////////////////////

__global__ void KernSdfSphere(BoundedVolume<SDF_t> vol, float3 center, float r)
{
    const int x = blockIdx.x*blockDim.x + threadIdx.x;
    const int y = blockIdx.y*blockDim.y + threadIdx.y;
    const int z = blockIdx.z*blockDim.z + threadIdx.z;

    const float3 pos = vol.VoxelPositionInUnits(x,y,z);
    const float dist = length(pos - center);
    const float sdf = dist - r;

    vol(x,y,z) = SDF_t(sdf);
}

void SdfSphere(BoundedVolume<SDF_t> vol, float3 center, float r)
{
    dim3 blockDim(8,8,8);
    dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);

    KernSdfSphere<<<gridDim,blockDim>>>(vol, center, r);
    GpuCheckErrors();
}

//////////////////////////////////////////////////////
// Take SDF Difference to depthmap
//////////////////////////////////////////////////////

__global__ void KernSdfDistance(Image<float> dist, Image<float> depth, BoundedVolume<SDF_t> vol, const Mat<float,3,4> T_wc, ImageIntrinsics K, float trunc_distance)
{
    const int u = blockIdx.x*blockDim.x + threadIdx.x;
    const int v = blockIdx.y*blockDim.y + threadIdx.y;
    
    if( u < depth.w && v < depth.h ) {
        const float z = depth(u,v);
        const float3 p_c = z * K.Unproject(u,v);
        const float3 p_w = T_wc * p_c;
        
        const SDF_t sdf = vol.GetUnitsTrilinearClamped(p_w);
        dist(u,v) = sdf.val; //(sdf.val + trunc_distance) / (2* trunc_distance);
    }    
}


void SdfDistance(Image<float> dist, Image<float> depth, BoundedVolume<SDF_t> vol, const Mat<float,3,4> T_wc, ImageIntrinsics K, float trunc_distance)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim, gridDim, depth);

    KernSdfDistance<<<gridDim,blockDim>>>(dist, depth, vol, T_wc, K, trunc_distance);
    GpuCheckErrors();
}

}
