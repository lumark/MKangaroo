#include "hip/hip_runtime.h"
#include "cu_sdffusion.h"

#include "MatUtils.h"
#include "launch_utils.h"

namespace roo
{
//////////////////////////////////////////////////////
/// Rolling SDF
//////////////////////////////////////////////////////

// Boxmin and boxmax define the box that is to be kept intact, rest will be cleared.
// This approach makes if conditions inside simpler.
// TODO: Name the function better.
__global__ void KernSdfResetPartial(BoundedVolume<SDF_t> vol, float3 boxmin, float3 boxmax)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;
  const int z = blockIdx.z*blockDim.z + threadIdx.z;

  const float3 P_w = vol.VoxelPositionInUnits(x,y,z);

  bool mincrit, maxcrit;//if mincrit and maxcrit are true, point is inside the box, i.e. valid.
  mincrit = P_w.x > boxmin.x && P_w.y < boxmax.y && P_w.z > boxmin.z;
  maxcrit = P_w.x < boxmax.x && P_w.y > boxmin.y && P_w.z < boxmax.z;

  if(!mincrit || !maxcrit)//i.e. the point is outside the box.
  {
    vol(x,y,z) = SDF_t(0.0/0.0,0.0);
  }
}


// TODO: Name the function better.
void SdfResetPartial(BoundedVolume<SDF_t> vol, float3 shift)
{
  //Initialization for GPU parallelization
  dim3 blockDim(8,8,8);
  dim3 gridDim(vol.w / blockDim.x, vol.h / blockDim.y, vol.d / blockDim.z);

  // compute the box to keep, it's conter intuitive to the name of function but
  // more efficient.
  float3 bn = vol.bbox.boxmin, bx = vol.bbox.boxmax;//bn is box min and bx is box max.

  if(shift.x>0)
    bn.x += shift.x;
  else
    bx.x += shift.x;

  // y is -ve, but boxmax and boxmin for y are also inverse. i.e. the bottom most
  // point is min.x,max.y,min.z
  if(shift.y>0)
    bn.y += shift.y;
  else
    bx.y += shift.y;

  if(shift.z>0)
    bn.z += shift.z;
  else
    bx.z += shift.z;

  KernSdfResetPartial<<<gridDim,blockDim>>>(vol, bn, bx);
  GpuCheckErrors();
}



//////////////////////////////////////////////////////
/// Rolling GRID SDF
//////////////////////////////////////////////////////

__device__ BoundedVolumeGrid<SDF_t, roo::TargetDevice, roo::Manage>  g_vol;
__device__ int                                                       g_NextResetSDFs[512];

// =============================================================================
// Boxmin and boxmax define the box that is to be kept intact, rest will be cleared.
// This approach makes if conditions inside simpler.
// When we clean a grid sdf, we also need to free its memory.. This maybe a little
// bit expensive
// =============================================================================

__global__ void KernRollingGridSdf(float3 boxmin, float3 boxmax, float3 shift)
{
  const int x = blockIdx.x*blockDim.x + threadIdx.x;
  const int y = blockIdx.y*blockDim.y + threadIdx.y;

  // For each voxel (x,y,z) we have in a bounded volume
  for(int z=0; z < g_vol.m_d; ++z)
  {
    const float3 P_w = g_vol.VoxelPositionInUnits(x,y,z);

    bool mincrit, maxcrit;//if mincrit and maxcrit are true, point is inside the box, i.e. valid.
    mincrit = P_w.x > boxmin.x && P_w.y > boxmin.y && P_w.z > boxmin.z;
    maxcrit = P_w.x < boxmax.x && P_w.y < boxmax.y && P_w.z < boxmax.z;

    if(!mincrit || !maxcrit)//i.e. the point is outside the box.
    {
      g_vol(x,y,z) = SDF_t(0.0/0.0,0.0);

      // get the index of grid sdf that need to be reseted
      int nIndex = int(floorf(x/g_vol.m_nVolumeGridRes)) +
          g_vol.m_nWholeGridRes * ( int(floorf(y/g_vol.m_nVolumeGridRes)) +
                                   g_vol.m_nWholeGridRes * int(floorf(z/g_vol.m_nVolumeGridRes)) );

      // save index of sdf that need to be reset later
      g_NextResetSDFs[nIndex] = 1;
    }

  }

}



// =============================================================================
// Rolling grid sdf. Each time we compute the index of sdf that we are going
// to reset and free the memory of it.
// the nature of rolling sdf is just shifting the index of sdf.
// shift is n * grid num in one diminsion. It can be positive or negative.
// by now each time we need to move at least one grid volume
// =============================================================================
void RollingGridSdfCuda(int* pNextInitSDFs, BoundedVolumeGrid<SDF_t> vol, int3 shift)
{
  hipMemcpyToSymbol(HIP_SYMBOL(g_vol), &vol, sizeof(vol), size_t(0), hipMemcpyHostToDevice);
  GpuCheckErrors();

  // 1, Compute the latest bounding box
  float3 bb_min = vol.m_bbox.boxmin, bb_max = vol.m_bbox.boxmax;

  if(shift.x!=0)
  {
    bb_min.x = bb_min.x + shift.x * vol.m_bbox.Size().x/float(vol.m_nWholeGridRes);
    bb_max.x = bb_max.x + shift.x * vol.m_bbox.Size().x/float(vol.m_nWholeGridRes);
  }

  if(shift.y!=0)
  {
    bb_min.y = bb_min.y + shift.y * vol.m_bbox.Size().y/float(vol.m_nWholeGridRes);
    bb_max.y = bb_max.y + shift.y * vol.m_bbox.Size().y/float(vol.m_nWholeGridRes);
  }

  if(shift.z!=0)
  {
    bb_min.z = bb_min.z + shift.z * vol.m_bbox.Size().z/float(vol.m_nWholeGridRes);
    bb_max.z = bb_max.z + shift.z * vol.m_bbox.Size().z/float(vol.m_nWholeGridRes);
  }

  // save shift params in grid sdf data struct
  vol.m_shift = shift;

  // 2, Kernel functin. Initialization for GPU parallelization
//  dim3 blockDim(16,16);
//  dim3 gridDim(vol.m_w / blockDim.x, vol.m_h / blockDim.y);
//  KernRollingGridSdf<<<gridDim,blockDim>>>(bb_min, bb_max, shift);
//  GpuCheckErrors();


  // 3, copy array back
  int nNextResetSDFs[vol.m_nWholeGridRes*vol.m_nWholeGridRes*vol.m_nWholeGridRes];
  hipMemcpyFromSymbol(nNextResetSDFs, HIP_SYMBOL(g_NextResetSDFs), sizeof(g_NextResetSDFs), 0, hipMemcpyDeviceToHost);
  GpuCheckErrors();

  for(int i=0;i!=vol.m_nWholeGridRes*vol.m_nWholeGridRes*vol.m_nWholeGridRes;i++)
  {
    pNextInitSDFs[i] = nNextResetSDFs[i];

    nNextResetSDFs[i] = 0;
  }

  // reset index
  hipMemcpyToSymbol(HIP_SYMBOL(g_NextResetSDFs),nNextResetSDFs,sizeof(nNextResetSDFs),0,hipMemcpyHostToDevice);

  // for each grid sdf that need to be reset, free it.
  g_vol.FreeMemory();

  // reset
  for(int i=0;i!=vol.m_nWholeGridRes*vol.m_nWholeGridRes*vol.m_nWholeGridRes;i++)
  {
    if(nNextResetSDFs[i]==1)
    {
      vol.FreeMemoryByIndex(i);
    }
  }
}

}
