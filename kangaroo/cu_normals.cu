#include "hip/hip_runtime.h"
#include "cu_normals.h"

#include "launch_utils.h"

namespace roo
{

//////////////////////////////////////////////////////
// Normals from VBO
//////////////////////////////////////////////////////

__global__ void KernNormalsFromVbo(Image<float4> dN, const Image<float4> dV)
{
    const int u = blockIdx.x*blockDim.x + threadIdx.x;
    const int v = blockIdx.y*blockDim.y + threadIdx.y;

    if( u < dN.w && v < dN.h) {
        if( u+1 < dN.w && v+1 < dN.h) {
            const float4 Vc = dV(u,v);
            const float4 Vr = dV(u+1,v);
            const float4 Vu = dV(u,v+1);
            const float4 a = Vr - Vc;
            const float4 b = Vu - Vc;

            const float3 axb = make_float3(
                a.y*b.z - a.z*b.y,
                a.z*b.x - a.x*b.z,
                a.x*b.y - a.y*b.x
            );

            const float magaxb = length(axb);
            const float4 N = make_float4(-axb.x/magaxb, -axb.y/magaxb, -axb.z/magaxb,1);
            dN(u,v) = N;
        }else{
            dN(u,v) = make_float4(0,0,0,0);
        }
    }
}

void NormalsFromVbo(Image<float4> dN, const Image<float4> dV)
{
    dim3 blockDim, gridDim;
    InitDimFromOutputImageOver(blockDim, gridDim, dN);
    KernNormalsFromVbo<<<gridDim,blockDim>>>(dN, dV);
}

}
